#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "dfwavelet_kernels.h"
#ifndef WAV_IMPL
#include "dfwavelet_impl.h"
#define WAV_IMPL
#endif
#  define _hdev_ __host__ __device__

// _data_t is the interal representation of data_t in CUDA
// Must be float2/double2 for data_t=Complex float/double or float/double for data_t=float/double
typedef double _data_t;

// Float2 Operators
inline _hdev_ float2 operator+ (float2 z1, float2 z2) {
  return make_float2 (z1.x + z2.x, z1.y + z2.y);		
}
inline _hdev_ float2 operator- (float2 z1, float2 z2) {
  return make_float2 (z1.x - z2.x, z1.y - z2.y);		
}
inline _hdev_ float2 operator* (float2 z1, float2 z2) {
  return make_float2 (z1.x*z2.x - z1.y*z2.y, z1.x*z2.y + z1.y*z2.x);		
}
inline _hdev_ float2 operator* (float2 z1, float alpha) {
  return make_float2 (z1.x*alpha, z1.y*alpha);		
}
inline _hdev_ float2 operator* (float alpha,float2 z1) {
  return make_float2 (z1.x*alpha, z1.y*alpha);		
}
inline _hdev_ void operator+= (float2 &z1, float2 z2) {
  z1.x += z2.x;
  z1.y += z2.y;		
}
inline _hdev_ float abs(float2 z1) {
  return sqrt(z1.x*z1.x + z1.y*z1.y);		
}

// Double2 Operators
inline _hdev_ double2 operator+ (double2 z1, double2 z2) {
  return make_double2 (z1.x + z2.x, z1.y + z2.y);		
}
inline _hdev_ double2 operator- (double2 z1, double2 z2) {
  return make_double2 (z1.x - z2.x, z1.y - z2.y);		
}
inline _hdev_ double2 operator* (double2 z1, double2 z2) {
  return make_double2 (z1.x*z2.x - z1.y*z2.y, z1.x*z2.y + z1.y*z2.x);		
}
inline _hdev_ double2 operator* (double2 z1, float alpha) {
  return make_double2 (z1.x*alpha, z1.y*alpha);		
}
inline _hdev_ double2 operator* (float alpha,double2 z1) {
  return make_double2 (z1.x*alpha, z1.y*alpha);		
}
inline _hdev_ void operator+= (double2 &z1, double2 z2) {
  z1.x += z2.x;
  z1.y += z2.y;		
}
inline _hdev_ float abs(double2 z1) {
  return sqrt(z1.x*z1.x + z1.y*z1.y);		
}

/********** Macros ************/
#define cuda(Call) do {					\
    hipError_t err = cuda ## Call ;			\
    if (err != hipSuccess){				\
      fprintf(stderr, "%s\n", hipGetErrorString(err));	\
      throw;						\
    }							\
  } while(0)

#define cuda_sync() do{				\
    cuda (ThreadSynchronize());			\
    cuda (GetLastError());			\
  } while(0)


/********** Macros ************/
#define cuda(Call) do {					\
    hipError_t err = cuda ## Call ;			\
    if (err != hipSuccess){				\
      fprintf(stderr, "%s\n", hipGetErrorString(err));	\
      throw;						\
    }							\
  } while(0)

#define cuda_sync() do{				\
    cuda (ThreadSynchronize());			\
    cuda (GetLastError());			\
  } while(0)

// ############################################################################
// Headers
// ############################################################################
static __global__ void cu_fwt3df_col(_data_t *Lx,_data_t *Hx,_data_t *in,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,scalar_t *lod,scalar_t *hid,int filterLen);
static __global__ void cu_fwt3df_row(_data_t *Ly,_data_t *Hy,_data_t *in,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,scalar_t *lod,scalar_t *hid,int filterLen);
static __global__ void cu_fwt3df_dep(_data_t *Lz,_data_t *Hz,_data_t *in,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,scalar_t *lod,scalar_t *hid,int filterLen);
static __global__ void cu_iwt3df_dep(_data_t *out,_data_t *Lz,_data_t *Hz,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,int xOffset,int yOffset,int zOffset,scalar_t *lod,scalar_t *hid,int filterLen);
static __global__ void cu_iwt3df_row(_data_t *out,_data_t *Ly,_data_t *Hy,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,int xOffset,int yOffset,int zOffset,scalar_t *lod,scalar_t *hid,int filterLen);
static __global__ void cu_iwt3df_col(_data_t *out,_data_t *Lx,_data_t *Hx,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,int xOffset,int yOffset,int zOffset,scalar_t *lod,scalar_t *hid,int filterLen);
static __global__ void cu_fwt3df_LC1(_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dxNext, int dyNext, int dzNext);
static __global__ void cu_fwt3df_LC2(_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dxNext, int dyNext, int dzNext);
static __global__ void cu_fwt3df_LC1_diff(_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dxNext, int dyNext, int dzNext);
static __global__ void cu_fwt3df_LC2_diff(_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dxNext, int dyNext, int dzNext);
static __global__ void cu_fwt3df_LC3(_data_t* HxHyHz_df1,_data_t* HxHyHz_df2,_data_t* HxHyHz_n,int dxNext, int dyNext, int dzNext);
static __global__ void cu_iwt3df_LC1(_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dx, int dy, int dz);
static __global__ void cu_iwt3df_LC2(_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dx, int dy, int dz);
static __global__ void cu_iwt3df_LC1_diff(_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dx, int dy, int dz);
static __global__ void cu_iwt3df_LC2_diff(_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dx, int dy, int dz);
static __global__ void cu_iwt3df_LC3(_data_t* HxHyHz_df1,_data_t* HxHyHz_df2,_data_t* HxHyHz_n,int dx, int dy, int dz);

static __global__ void cu_add(_data_t* out, _data_t* in, int maxInd);
static __global__ void cu_mult(_data_t* in, _data_t mult, int maxInd);
static __global__ void cu_soft_thresh (_data_t* in, scalar_t thresh, int numMax);
static __global__ void cu_circshift(_data_t* data, _data_t* dataCopy, int dx, int dy, int dz, int shift1, int shift2, int shift3);
static __global__ void cu_circunshift(_data_t* data, _data_t* dataCopy, int dx, int dy, int dz, int shift1, int shift2, int shift3);

void dfSUREshrink_gpu(struct dfwavelet_plan_s* plan,scalar_t sigma,data_t* dev_wcdf1,data_t* dev_wcdf2,data_t* dev_wcn);
scalar_t get_SURE_thresh (scalar_t* subband2, scalar_t* cum_sum,scalar_t sigma2,int length);
void prefix_sum(scalar_t* ener,scalar_t* out_values, scalar_t *in_values,int length);
void count_zeros_gpu (struct dfwavelet_plan_s* plan,data_t* in_vx,data_t* in_vy,data_t* in_vz);
void bitonic_sort(scalar_t *dev_values,int length);
scalar_t getMADsigma_gpu(struct dfwavelet_plan_s* plan, data_t* wcn);

static __global__ void cu_subband2 (scalar_t* subband2, data_t* subband,int length);
static __global__ void cu_calc_SURE_risk (scalar_t* risk, scalar_t* subband2, scalar_t* cum_sum,scalar_t sigma2,int length);
static __global__ void cu_find_min_risk (scalar_t* out_risk,scalar_t* out_subband2,scalar_t* in_risk,scalar_t* in_subband2,int length);
static __global__ void cu_is_zeros (scalar_t* isZeros,data_t* in_vx,data_t* in_vy,data_t* in_vz,int length);
static __global__ void cu_sumAll (scalar_t* out,scalar_t* in, int length);
static __global__ void cu_bitonic_sort_step(scalar_t *dev_values, int j, int k);
static __global__ void cu_prescan(scalar_t* out,scalar_t *in, int n)  ;
static __global__ void cu_add_cum(scalar_t *out,scalar_t* in);


extern "C" void dffwt3_gpuHost(struct dfwavelet_plan_s* plan, data_t* out_wcdf1,data_t* out_wcdf2,data_t* out_wcn, data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  assert(plan->use_gpu==2);
  data_t* dev_wcdf1,*dev_wcdf2,*dev_wcn,*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  cuda(Memcpy( dev_vx, in_vx, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vy, in_vy, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vz, in_vz, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));

  cuda(Malloc( (void**)&dev_wcdf1, plan->numCoeff*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_wcdf2, plan->numCoeff*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_wcn, plan->numCoeff*sizeof(data_t) ));

  dffwt3_gpu(plan,dev_wcdf1,dev_wcdf2,dev_wcn,dev_vx,dev_vy,dev_vz);

  cuda(Memcpy( out_wcdf1, dev_wcdf1, plan->numCoeff*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_wcdf2, dev_wcdf2, plan->numCoeff*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_wcn, dev_wcn, plan->numCoeff*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_wcdf1 ));
  cuda(Free( dev_wcdf2 ));
  cuda(Free( dev_wcn ));
  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dfiwt3_gpuHost(struct dfwavelet_plan_s* plan, data_t* out_vx,data_t* out_vy,data_t* out_vz, data_t* in_wcdf1,data_t* in_wcdf2,data_t* in_wcn)
{
  assert(plan->use_gpu==2);
  data_t* dev_wcdf1,*dev_wcdf2,*dev_wcn,*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_wcdf1, plan->numCoeff*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_wcdf2, plan->numCoeff*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_wcn, plan->numCoeff*sizeof(data_t) ));

  cuda(Memcpy( dev_wcdf1, in_wcdf1, plan->numCoeff*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_wcdf2, in_wcdf2, plan->numCoeff*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_wcn, in_wcn, plan->numCoeff*sizeof(data_t), hipMemcpyHostToDevice ));

  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  dfiwt3_gpu(plan,dev_vx,dev_vy,dev_vz,dev_wcdf1,dev_wcdf2,dev_wcn);
  cuda(Memcpy( out_vx, dev_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vy, dev_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vz, dev_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_wcdf1 ));
  cuda(Free( dev_wcdf2 ));
  cuda(Free( dev_wcn ));
  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dfsoftthresh_gpuHost(struct dfwavelet_plan_s* plan,scalar_t dfthresh, scalar_t nthresh, data_t* out_wcdf1,data_t* out_wcdf2,data_t* out_wcn)
{
  assert(plan->use_gpu==2);
  data_t* dev_wcdf1,*dev_wcdf2,*dev_wcn;
  cuda(Malloc( (void**)&dev_wcdf1, plan->numCoeff*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_wcdf2, plan->numCoeff*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_wcn, plan->numCoeff*sizeof(data_t) ));

  cuda(Memcpy( dev_wcdf1, out_wcdf1, plan->numCoeff*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_wcdf2, out_wcdf2, plan->numCoeff*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_wcn, out_wcn, plan->numCoeff*sizeof(data_t), hipMemcpyHostToDevice ));

  dfsoftthresh_gpu(plan,dfthresh,nthresh,dev_wcdf1,dev_wcdf2,dev_wcn);

  cuda(Memcpy( out_wcdf1, dev_wcdf1, plan->numCoeff*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_wcdf2, dev_wcdf2, plan->numCoeff*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_wcn, dev_wcn, plan->numCoeff*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_wcdf1 ));
  cuda(Free( dev_wcdf2 ));
  cuda(Free( dev_wcn ));
}

extern "C" void dfwavthresh3_gpuHost(struct dfwavelet_plan_s* plan, scalar_t dfthresh,scalar_t nthresh,data_t* out_vx,data_t* out_vy,data_t* out_vz, data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  assert(plan->use_gpu==2);
  data_t*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  cuda(Memcpy( dev_vx, in_vx, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vy, in_vy, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vz, in_vz, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));

  dfwavthresh3_gpu(plan,dfthresh,nthresh,dev_vx,dev_vy,dev_vz,dev_vx,dev_vy,dev_vz);

  cuda(Memcpy( out_vx, dev_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vy, dev_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vz, dev_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dfwavthresh3_spin_gpuHost(struct dfwavelet_plan_s* plan, scalar_t dfthresh,scalar_t nthresh,int spins,int isRand,data_t* out_vx,data_t* out_vy,data_t* out_vz, data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  assert(plan->use_gpu==2);
  data_t*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  cuda(Memcpy( dev_vx, in_vx, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vy, in_vy, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vz, in_vz, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));

  dfwavthresh3_spin_gpu(plan,dfthresh,nthresh,spins,isRand,dev_vx,dev_vy,dev_vz,dev_vx,dev_vy,dev_vz);

  cuda(Memcpy( out_vx, dev_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vy, dev_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vz, dev_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dfwavthresh3_SURE_gpuHost(struct dfwavelet_plan_s* plan,scalar_t sigma,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  assert(plan->use_gpu==2);
  data_t*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  cuda(Memcpy( dev_vx, in_vx, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vy, in_vy, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vz, in_vz, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));

  dfwavthresh3_SURE_gpu(plan,sigma,dev_vx,dev_vy,dev_vz,dev_vx,dev_vy,dev_vz);

  cuda(Memcpy( out_vx, dev_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vy, dev_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vz, dev_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dfwavthresh3_SURE_spin_gpuHost(struct dfwavelet_plan_s* plan,scalar_t sigma,int spins,int isRand,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  assert(plan->use_gpu==2);
  data_t*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  cuda(Memcpy( dev_vx, in_vx, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vy, in_vy, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vz, in_vz, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));

  dfwavthresh3_SURE_spin_gpu(plan,sigma,spins,isRand,dev_vx,dev_vy,dev_vz,dev_vx,dev_vy,dev_vz);

  cuda(Memcpy( out_vx, dev_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vy, dev_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vz, dev_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dfwavthresh3_SURE_MAD_gpuHost(struct dfwavelet_plan_s* plan,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  assert(plan->use_gpu==2);
  data_t*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  cuda(Memcpy( dev_vx, in_vx, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vy, in_vy, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vz, in_vz, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));

  dfwavthresh3_SURE_MAD_gpu(plan,dev_vx,dev_vy,dev_vz,dev_vx,dev_vy,dev_vz);

  cuda(Memcpy( out_vx, dev_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vy, dev_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vz, dev_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dfwavthresh3_SURE_MAD_spin_gpuHost(struct dfwavelet_plan_s* plan,int spins,int isRand,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  assert(plan->use_gpu==2);
  data_t*dev_vx,*dev_vy,*dev_vz;
  cuda(Malloc( (void**)&dev_vx, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vy, plan->numPixel*sizeof(data_t) ));
  cuda(Malloc( (void**)&dev_vz, plan->numPixel*sizeof(data_t) ));

  cuda(Memcpy( dev_vx, in_vx, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vy, in_vy, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_vz, in_vz, plan->numPixel*sizeof(data_t), hipMemcpyHostToDevice ));

  dfwavthresh3_SURE_MAD_spin_gpu(plan,spins,isRand,dev_vx,dev_vy,dev_vz,dev_vx,dev_vy,dev_vz);

  cuda(Memcpy( out_vx, dev_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vy, dev_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));
  cuda(Memcpy( out_vz, dev_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToHost ));

  cuda(Free( dev_vx ));
  cuda(Free( dev_vy ));
  cuda(Free( dev_vz ));
}

extern "C" void dffwt3_gpu(struct dfwavelet_plan_s* plan, data_t* out_wcdf1,data_t* out_wcdf2,data_t* out_wcn, data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  circshift_gpu(plan,in_vx);
  circshift_gpu(plan,in_vy);
  circshift_gpu(plan,in_vz);
  
  int numCoeff, filterLen,*waveSizes,numLevels;
  numCoeff = plan->numCoeff;
  waveSizes = plan->waveSizes;
  filterLen = plan->filterLen;
  numLevels = plan->numLevels;
  // Cast from generic data_t to device compatible _data_t
  _data_t* dev_wcdf1 = (data_t*) out_wcdf1;
  _data_t* dev_wcdf2 = (data_t*) out_wcdf2;
  _data_t* dev_wcn = (data_t*) out_wcn;
  _data_t* dev_in_vx = (data_t*) in_vx;
  _data_t* dev_in_vy = (data_t*) in_vy;
  _data_t* dev_in_vz = (data_t*) in_vz;
  _data_t* dev_temp1,*dev_temp2;
  cuda(Malloc( (void**)&dev_temp1, numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_temp2, numCoeff*sizeof(_data_t) ));

  // Get dimensions
  int dx = plan->imSize[0];
  int dy = plan->imSize[1];
  int dz = plan->imSize[2];
  int dxNext = waveSizes[0 + 3*numLevels];
  int dyNext = waveSizes[1 + 3*numLevels];
  int dzNext = waveSizes[2 + 3*numLevels];
  int blockSize = dxNext*dyNext*dzNext;

  // allocate device memory and  copy filters to device
  scalar_t *dev_filters;
  cuda(Malloc( (void**)&dev_filters, 4*plan->filterLen*sizeof(scalar_t) ));
  scalar_t *dev_lod0 = dev_filters + 0*plan->filterLen;
  scalar_t *dev_hid0 = dev_filters + 1*plan->filterLen;
  scalar_t *dev_lod1 = dev_filters + 2*plan->filterLen;
  scalar_t *dev_hid1 = dev_filters + 3*plan->filterLen;
  cuda(Memcpy( dev_lod0, plan->lod0, 2*plan->filterLen*sizeof(scalar_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_lod1, plan->lod1, 2*plan->filterLen*sizeof(scalar_t), hipMemcpyHostToDevice ));

  // Initialize variables and Pointers for FWT
  int const SHMEM_SIZE = 16384;
  int const T = 512;
  int mem, K;
  dim3 numBlocks, numThreads;

  // Temp Pointers
  _data_t *dev_tempLx,*dev_tempHx;
  dev_tempLx = dev_temp1;
  dev_tempHx = dev_tempLx + numCoeff/2;
  _data_t *dev_tempLxLy,*dev_tempHxLy,*dev_tempLxHy,*dev_tempHxHy;
  dev_tempLxLy = dev_temp2;
  dev_tempHxLy = dev_tempLxLy + numCoeff/4;
  dev_tempLxHy = dev_tempHxLy + numCoeff/4;
  dev_tempHxHy = dev_tempLxHy + numCoeff/4;

  // wcdf1 Pointers
  _data_t *dev_LxLyLz_df1,*dev_HxLyLz_df1,*dev_LxHyLz_df1,*dev_HxHyLz_df1,*dev_LxLyHz_df1,*dev_HxLyHz_df1,*dev_LxHyHz_df1,*dev_HxHyHz_df1,*dev_current_vx;
  dev_LxLyLz_df1 = dev_wcdf1;
  dev_HxLyLz_df1 = dev_LxLyLz_df1 + waveSizes[0]*waveSizes[1]*waveSizes[2];
  for (int l = 1; l <= numLevels; ++l){
    dev_HxLyLz_df1 += 7*waveSizes[0 + 3*l]*waveSizes[1 + 3*l]*waveSizes[2 + 3*l];
  }
  dev_current_vx = dev_in_vx;

  // wcdf2 Pointers
  _data_t *dev_LxLyLz_df2,*dev_HxLyLz_df2,*dev_LxHyLz_df2,*dev_HxHyLz_df2,*dev_LxLyHz_df2,*dev_HxLyHz_df2,*dev_LxHyHz_df2,*dev_HxHyHz_df2,*dev_current_vy;
  dev_LxLyLz_df2 = dev_wcdf2;
  dev_HxLyLz_df2 = dev_LxLyLz_df2 + waveSizes[0]*waveSizes[1]*waveSizes[2];
  for (int l = 1; l <= numLevels; ++l){
    dev_HxLyLz_df2 += 7*waveSizes[0 + 3*l]*waveSizes[1 + 3*l]*waveSizes[2 + 3*l];
  }
  dev_current_vy = dev_in_vy;

  // wcn Pointers
  _data_t *dev_LxLyLz_n,*dev_HxLyLz_n,*dev_LxHyLz_n,*dev_HxHyLz_n,*dev_LxLyHz_n,*dev_HxLyHz_n,*dev_LxHyHz_n,*dev_HxHyHz_n,*dev_current_vz;
  dev_LxLyLz_n = dev_wcn;
  dev_HxLyLz_n = dev_LxLyLz_n + waveSizes[0]*waveSizes[1]*waveSizes[2];
  for (int l = 1; l <= numLevels; ++l){
    dev_HxLyLz_n += 7*waveSizes[0 + 3*l]*waveSizes[1 + 3*l]*waveSizes[2 + 3*l];
  }
  dev_current_vz = dev_in_vz;

  //*****************Loop through levels****************
  for (int l = numLevels; l >= 1; --l)
    {
      dxNext = waveSizes[0 + 3*l];
      dyNext = waveSizes[1 + 3*l];
      dzNext = waveSizes[2 + 3*l];
      blockSize = dxNext*dyNext*dzNext;

      // Update Pointers
      // df1
      dev_HxLyLz_df1 = dev_HxLyLz_df1 - 7*blockSize;
      dev_LxHyLz_df1 = dev_HxLyLz_df1 + blockSize;
      dev_HxHyLz_df1 = dev_LxHyLz_df1 + blockSize;
      dev_LxLyHz_df1 = dev_HxHyLz_df1 + blockSize;
      dev_HxLyHz_df1 = dev_LxLyHz_df1 + blockSize;
      dev_LxHyHz_df1 = dev_HxLyHz_df1 + blockSize;
      dev_HxHyHz_df1 = dev_LxHyHz_df1 + blockSize;
      // df2
      dev_HxLyLz_df2 = dev_HxLyLz_df2 - 7*blockSize;
      dev_LxHyLz_df2 = dev_HxLyLz_df2 + blockSize;
      dev_HxHyLz_df2 = dev_LxHyLz_df2 + blockSize;
      dev_LxLyHz_df2 = dev_HxHyLz_df2 + blockSize;
      dev_HxLyHz_df2 = dev_LxLyHz_df2 + blockSize;
      dev_LxHyHz_df2 = dev_HxLyHz_df2 + blockSize;
      dev_HxHyHz_df2 = dev_LxHyHz_df2 + blockSize;
      // n
      dev_HxLyLz_n = dev_HxLyLz_n - 7*blockSize;
      dev_LxHyLz_n = dev_HxLyLz_n + blockSize;
      dev_HxHyLz_n = dev_LxHyLz_n + blockSize;
      dev_LxLyHz_n = dev_HxHyLz_n + blockSize;
      dev_HxLyHz_n = dev_LxLyHz_n + blockSize;
      dev_LxHyHz_n = dev_HxLyHz_n + blockSize;
      dev_HxHyHz_n = dev_LxHyHz_n + blockSize;

      //************WCVX***********
      // FWT Columns
      K = (SHMEM_SIZE-16)/(dx*sizeof(_data_t));
      numBlocks = dim3(1,(dy+K-1)/K,dz);
      numThreads = dim3(T/K,K,1);
      mem = K*dx*sizeof(_data_t);

      cu_fwt3df_col <<< numBlocks,numThreads,mem >>>(dev_tempLx,dev_tempHx,dev_current_vx,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod1,dev_hid1,filterLen);
      cuda_sync();
      // FWT Rows
      K = (SHMEM_SIZE-16)/(dy*sizeof(_data_t));
      numBlocks = dim3(((dxNext)+K-1)/K,1,dz);
      numThreads = dim3(K,T/K,1);
      mem = K*dy*sizeof(_data_t);
      
      cu_fwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempLxLy,dev_tempLxHy,dev_tempLx,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempHxLy,dev_tempHxHy,dev_tempHx,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cuda_sync();
      // FWT Depths
      K = (SHMEM_SIZE-16)/(dz*sizeof(_data_t));
      numBlocks = dim3(((dxNext)+K-1)/K,dyNext,1);
      numThreads = dim3(K,1,T/K);
      mem = K*dz*sizeof(_data_t);
      
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_LxLyLz_df1,dev_LxLyHz_df1,dev_tempLxLy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_LxHyLz_df1,dev_LxHyHz_df1,dev_tempLxHy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_HxLyLz_df1,dev_HxLyHz_df1,dev_tempHxLy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_HxHyLz_df1,dev_HxHyHz_df1,dev_tempHxHy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cuda_sync();

      //************WCVY***********
      // FWT Columns
      K = (SHMEM_SIZE-16)/(dx*sizeof(_data_t));
      numBlocks = dim3(1,(dy+K-1)/K,dz);
      numThreads = dim3(T/K,K,1);
      mem = K*dx*sizeof(_data_t);
      
      cu_fwt3df_col <<< numBlocks,numThreads,mem >>>(dev_tempLx,dev_tempHx,dev_current_vy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cuda_sync();
      // FWT Rows
      K = (SHMEM_SIZE-16)/(dy*sizeof(_data_t));
      numBlocks = dim3(((dxNext)+K-1)/K,1,dz);
      numThreads = dim3(K,T/K,1);
      mem = K*dy*sizeof(_data_t);
      
      cu_fwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempLxLy,dev_tempLxHy,dev_tempLx,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod1,dev_hid1,filterLen);
      cu_fwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempHxLy,dev_tempHxHy,dev_tempHx,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod1,dev_hid1,filterLen);
      cuda_sync();
      // FWT Depths
      K = (SHMEM_SIZE-16)/(dz*sizeof(_data_t));
      numBlocks = dim3(((dxNext)+K-1)/K,dyNext,1);
      numThreads = dim3(K,1,T/K);
      mem = K*dz*sizeof(_data_t);
      
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_LxLyLz_df2,dev_LxLyHz_df2,dev_tempLxLy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_LxHyLz_df2,dev_LxHyHz_df2,dev_tempLxHy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_HxLyLz_df2,dev_HxLyHz_df2,dev_tempHxLy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_HxHyLz_df2,dev_HxHyHz_df2,dev_tempHxHy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cuda_sync();

      //************WCVZ***********
      // FWT Columns
      K = (SHMEM_SIZE-16)/(dx*sizeof(_data_t));
      numBlocks = dim3(1,(dy+K-1)/K,dz);
      numThreads = dim3(T/K,K,1);
      mem = K*dx*sizeof(_data_t);

      cu_fwt3df_col <<< numBlocks,numThreads,mem >>>(dev_tempLx,dev_tempHx,dev_current_vz,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cuda_sync();
      // FWT Rows
      K = (SHMEM_SIZE-16)/(dy*sizeof(_data_t));
      numBlocks = dim3(((dxNext)+K-1)/K,1,dz);
      numThreads = dim3(K,T/K,1);
      mem = K*dy*sizeof(_data_t);
      
      cu_fwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempLxLy,dev_tempLxHy,dev_tempLx,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cu_fwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempHxLy,dev_tempHxHy,dev_tempHx,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod0,dev_hid0,filterLen);
      cuda_sync();
      // FWT Depths
      K = (SHMEM_SIZE-16)/(dz*sizeof(_data_t));
      numBlocks = dim3(((dxNext)+K-1)/K,dyNext,1);
      numThreads = dim3(K,1,T/K);
      mem = K*dz*sizeof(_data_t);
      
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_LxLyLz_n,dev_LxLyHz_n,dev_tempLxLy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod1,dev_hid1,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_LxHyLz_n,dev_LxHyHz_n,dev_tempLxHy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod1,dev_hid1,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_HxLyLz_n,dev_HxLyHz_n,dev_tempHxLy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod1,dev_hid1,filterLen);
      cu_fwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_HxHyLz_n,dev_HxHyHz_n,dev_tempHxHy,dx,dy,dz,dxNext,dyNext,dzNext,dev_lod1,dev_hid1,filterLen);
      cuda_sync();

      //******* Multi ******
      int maxInd = 7*blockSize;
      numThreads = T;
      numBlocks = (maxInd+numThreads.x-1)/numThreads.x;
      cu_mult <<< numBlocks, numThreads >>> (dev_HxLyLz_df1,1./plan->res[0],maxInd);
      cu_mult <<< numBlocks, numThreads >>> (dev_HxLyLz_df2,1./plan->res[1],maxInd);
      cu_mult <<< numBlocks, numThreads >>> (dev_HxLyLz_n,1./plan->res[2],maxInd);
      cuda_sync();

      //*******Linear Combination******
      int t1 = min(dxNext,T);
      int t2 = T/t1;
      numBlocks = dim3( (dxNext+t1-1)/t1, (dyNext+t2-1)/t2, dzNext);
      numThreads = dim3(t1,t2,1);
	  
      cu_fwt3df_LC1 <<< numBlocks,numThreads >>> (dev_HxLyLz_df1,dev_HxLyLz_df2,dev_HxLyLz_n,dev_LxHyLz_df1,dev_LxHyLz_df2,dev_LxHyLz_n,dev_LxLyHz_df1,dev_LxLyHz_df2,dev_LxLyHz_n,dxNext,dyNext,dzNext);
      cu_fwt3df_LC2 <<< numBlocks,numThreads >>> (dev_HxHyLz_df1,dev_HxHyLz_df2,dev_HxHyLz_n,dev_HxLyHz_df1,dev_HxLyHz_df2,dev_HxLyHz_n,dev_LxHyHz_df1,dev_LxHyHz_df2,dev_LxHyHz_n,dxNext,dyNext,dzNext);
      cu_fwt3df_LC3 <<< numBlocks,numThreads >>> (dev_HxHyHz_df1,dev_HxHyHz_df2,dev_HxHyHz_n,dxNext,dyNext,dzNext);
      cuda_sync();
      cu_fwt3df_LC1_diff <<< numBlocks,numThreads >>> (dev_HxLyLz_df1,dev_HxLyLz_df2,dev_HxLyLz_n,dev_LxHyLz_df1,dev_LxHyLz_df2,dev_LxHyLz_n,dev_LxLyHz_df1,dev_LxLyHz_df2,dev_LxLyHz_n,dxNext,dyNext,dzNext);
      cu_fwt3df_LC2_diff <<< numBlocks,numThreads >>> (dev_HxHyLz_df1,dev_HxHyLz_df2,dev_HxHyLz_n,dev_HxLyHz_df1,dev_HxLyHz_df2,dev_HxLyHz_n,dev_LxHyHz_df1,dev_LxHyHz_df2,dev_LxHyHz_n,dxNext,dyNext,dzNext);
      cuda_sync();

      dev_current_vx = dev_wcdf1;
      dev_current_vy = dev_wcdf2;
      dev_current_vz = dev_wcn;

      dx = dxNext;
      dy = dyNext;
      dz = dzNext;
    }
  cuda(Free( dev_filters ));
  cuda(Free( dev_temp1 ));
  cuda(Free( dev_temp2 ));
  
  circunshift_gpu(plan,in_vx);
  circunshift_gpu(plan,in_vy);
  circunshift_gpu(plan,in_vz);
}

extern "C" void dfiwt3_gpu(struct dfwavelet_plan_s* plan, data_t* out_vx,data_t* out_vy,data_t* out_vz, data_t* in_wcdf1,data_t* in_wcdf2,data_t* in_wcn)
{
  int numCoeff, filterLen,*waveSizes,numLevels;
  numCoeff = plan->numCoeff;
  waveSizes = plan->waveSizes;
  filterLen = plan->filterLen;
  numLevels = plan->numLevels;
  // Cast from generic data_t to device compatible _data_t
  data_t* dev_out_vx = (_data_t*)out_vx;
  data_t* dev_out_vy = (_data_t*)out_vy;
  data_t* dev_out_vz = (_data_t*)out_vz;
  data_t* dev_wcdf1 = (_data_t*)in_wcdf1;
  data_t* dev_wcdf2 = (_data_t*)in_wcdf2;
  data_t* dev_wcn = (_data_t*)in_wcn;
  _data_t* dev_temp1, *dev_temp2;
  cuda(Malloc( (void**)&dev_temp1, numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_temp2, numCoeff*sizeof(_data_t)) );
  // allocate device memory
  scalar_t *dev_filters;
  cuda(Malloc( (void**)&dev_filters, 4*(plan->filterLen)*sizeof(scalar_t) ));
  scalar_t *dev_lor0 = dev_filters + 0*plan->filterLen;
  scalar_t *dev_hir0 = dev_filters + 1*plan->filterLen;
  scalar_t *dev_lor1 = dev_filters + 2*plan->filterLen;
  scalar_t *dev_hir1 = dev_filters + 3*plan->filterLen;
  cuda(Memcpy( dev_lor0, plan->lor0, 2*plan->filterLen*sizeof(scalar_t), hipMemcpyHostToDevice ));
  cuda(Memcpy( dev_lor1, plan->lor1, 2*plan->filterLen*sizeof(scalar_t), hipMemcpyHostToDevice ));
      
  // Workspace dimensions
  int dxWork = waveSizes[0 + 3*numLevels]*2-1 + filterLen-1;
  int dyWork = waveSizes[1 + 3*numLevels]*2-1 + filterLen-1;
  int dzWork = waveSizes[2 + 3*numLevels]*2-1 + filterLen-1;

  // Initialize variables and pointers for IWT
  int const SHMEM_SIZE = 16384;
  int const T = 512;
  int mem,K;
  dim3 numBlocks, numThreads;
  int dx = waveSizes[0];
  int dy = waveSizes[1];
  int dz = waveSizes[2];

  // Temp Pointers
  _data_t *dev_tempLxLy,*dev_tempHxLy,*dev_tempLxHy,*dev_tempHxHy;
  dev_tempLxLy = dev_temp1;
  dev_tempHxLy = dev_tempLxLy + numCoeff/4;
  dev_tempLxHy = dev_tempHxLy + numCoeff/4;
  dev_tempHxHy = dev_tempLxHy + numCoeff/4;
  _data_t *dev_tempLx,*dev_tempHx;
  dev_tempLx = dev_temp2;
  dev_tempHx = dev_tempLx + numCoeff/2;
  // wcdf1 Pointers
  _data_t *dev_LxLyLz_df1,*dev_HxLyLz_df1,*dev_LxHyLz_df1,*dev_HxHyLz_df1,*dev_LxLyHz_df1,*dev_HxLyHz_df1,*dev_LxHyHz_df1,*dev_HxHyHz_df1,*dev_current_vx;
  dev_LxLyLz_df1 = dev_wcdf1;
  dev_HxLyLz_df1 = dev_LxLyLz_df1 + dx*dy*dz;
  dev_current_vx = dev_LxLyLz_df1;
  // wcdf2 Pointers
  _data_t *dev_LxLyLz_df2,*dev_HxLyLz_df2,*dev_LxHyLz_df2,*dev_HxHyLz_df2,*dev_LxLyHz_df2,*dev_HxLyHz_df2,*dev_LxHyHz_df2,*dev_HxHyHz_df2,*dev_current_vy;
  dev_LxLyLz_df2 = dev_wcdf2;
  dev_HxLyLz_df2 = dev_LxLyLz_df2 + dx*dy*dz;
  dev_current_vy = dev_LxLyLz_df2;
  // wcn Pointers
  _data_t *dev_LxLyLz_n,*dev_HxLyLz_n,*dev_LxHyLz_n,*dev_HxHyLz_n,*dev_LxLyHz_n,*dev_HxLyHz_n,*dev_LxHyHz_n,*dev_HxHyHz_n,*dev_current_vz;
  dev_LxLyLz_n = dev_wcn;
  dev_HxLyLz_n = dev_LxLyLz_n + dx*dy*dz;
  dev_current_vz = dev_LxLyLz_n;

  for (int level = 1; level < numLevels+1; ++level)
    {
      dx = waveSizes[0 + 3*level];
      dy = waveSizes[1 + 3*level];
      dz = waveSizes[2 + 3*level];
      int blockSize = dx*dy*dz;
      int dxNext = waveSizes[0+3*(level+1)];
      int dyNext = waveSizes[1+3*(level+1)];
      int dzNext = waveSizes[2+3*(level+1)];
	  
      // Calclate Offset
      dxWork = (2*dx-1 + filterLen-1);
      dyWork = (2*dy-1 + filterLen-1);
      dzWork = (2*dz-1 + filterLen-1);
      int xOffset = (int) floor((dxWork - dxNext) / 2.0);
      int yOffset = (int) floor((dyWork - dyNext) / 2.0);
      int zOffset = (int) floor((dzWork - dzNext) / 2.0);

      // Update Pointers
      // df1
      dev_LxHyLz_df1 = dev_HxLyLz_df1 + blockSize;
      dev_HxHyLz_df1 = dev_LxHyLz_df1 + blockSize;
      dev_LxLyHz_df1 = dev_HxHyLz_df1 + blockSize;
      dev_HxLyHz_df1 = dev_LxLyHz_df1 + blockSize;
      dev_LxHyHz_df1 = dev_HxLyHz_df1 + blockSize;
      dev_HxHyHz_df1 = dev_LxHyHz_df1 + blockSize;
      // df2
      dev_LxHyLz_df2 = dev_HxLyLz_df2 + blockSize;
      dev_HxHyLz_df2 = dev_LxHyLz_df2 + blockSize;
      dev_LxLyHz_df2 = dev_HxHyLz_df2 + blockSize;
      dev_HxLyHz_df2 = dev_LxLyHz_df2 + blockSize;
      dev_LxHyHz_df2 = dev_HxLyHz_df2 + blockSize;
      dev_HxHyHz_df2 = dev_LxHyHz_df2 + blockSize;
      // n
      dev_LxHyLz_n = dev_HxLyLz_n + blockSize;
      dev_HxHyLz_n = dev_LxHyLz_n + blockSize;
      dev_LxLyHz_n = dev_HxHyLz_n + blockSize;
      dev_HxLyHz_n = dev_LxLyHz_n + blockSize;
      dev_LxHyHz_n = dev_HxLyHz_n + blockSize;
      dev_HxHyHz_n = dev_LxHyHz_n + blockSize;

      //*******Linear Combination******

      int t1 = min(dxNext,T);
      int t2 = T/t1;
      numBlocks = dim3( (dx+t1-1)/t1, (dy+t2-1)/t2, dz);
      numThreads = dim3(t1,t2,1);

      cu_iwt3df_LC1 <<< numBlocks,numThreads >>> (dev_HxLyLz_df1,dev_HxLyLz_df2,dev_HxLyLz_n,dev_LxHyLz_df1,dev_LxHyLz_df2,dev_LxHyLz_n,dev_LxLyHz_df1,dev_LxLyHz_df2,dev_LxLyHz_n,dx,dy,dz);
      cu_iwt3df_LC2 <<< numBlocks,numThreads >>> (dev_HxHyLz_df1,dev_HxHyLz_df2,dev_HxHyLz_n,dev_HxLyHz_df1,dev_HxLyHz_df2,dev_HxLyHz_n,dev_LxHyHz_df1,dev_LxHyHz_df2,dev_LxHyHz_n,dx,dy,dz);
      cu_iwt3df_LC3 <<< numBlocks,numThreads >>> (dev_HxHyHz_df1,dev_HxHyHz_df2,dev_HxHyHz_n,dx,dy,dz);
      cuda_sync();
      cu_iwt3df_LC1_diff <<< numBlocks,numThreads >>> (dev_HxLyLz_df1,dev_HxLyLz_df2,dev_HxLyLz_n,dev_LxHyLz_df1,dev_LxHyLz_df2,dev_LxHyLz_n,dev_LxLyHz_df1,dev_LxLyHz_df2,dev_LxLyHz_n,dx,dy,dz);
      cu_iwt3df_LC2_diff <<< numBlocks,numThreads >>> (dev_HxHyLz_df1,dev_HxHyLz_df2,dev_HxHyLz_n,dev_HxLyHz_df1,dev_HxLyHz_df2,dev_HxLyHz_n,dev_LxHyHz_df1,dev_LxHyHz_df2,dev_LxHyHz_n,dx,dy,dz);
      cuda_sync();
      
      //******* Multi ******
      int maxInd = 7*blockSize;
      numThreads = T;
      numBlocks = (maxInd+numThreads.x-1)/numThreads.x;
      cu_mult <<< numBlocks, numThreads >>> (dev_HxLyLz_df1,plan->res[0],maxInd);
      cu_mult <<< numBlocks, numThreads >>> (dev_HxLyLz_df2,plan->res[1],maxInd);
      cu_mult <<< numBlocks, numThreads >>> (dev_HxLyLz_n,plan->res[2],maxInd);
      cuda_sync();

      //************WCX************
      // Update Pointers
      if (level==numLevels)
	dev_current_vx = dev_out_vx;
      // IWT Depths
      K = (SHMEM_SIZE-16)/(2*dz*sizeof(_data_t));
      numBlocks = dim3((dx+K-1)/K,dy,1);
      numThreads = dim3(K,1,(T/K));
      mem = K*2*dz*sizeof(_data_t);

      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempLxLy,dev_LxLyLz_df1,dev_LxLyHz_df1,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempHxLy,dev_HxLyLz_df1,dev_HxLyHz_df1,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempLxHy,dev_LxHyLz_df1,dev_LxHyHz_df1,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempHxHy,dev_HxHyLz_df1,dev_HxHyHz_df1,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cuda_sync();
      // IWT Rows
      K = (SHMEM_SIZE-16)/(2*dy*sizeof(_data_t));
      numBlocks = dim3((dx+K-1)/K,1,dzNext);
      numThreads = dim3(K,(T/K),1);
      mem = K*2*dy*sizeof(_data_t);

      cu_iwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempLx,dev_tempLxLy,dev_tempLxHy,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,plan->filterLen);
      cu_iwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempHx,dev_tempHxLy,dev_tempHxHy,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,plan->filterLen);
      cuda_sync();
      // IWT Columns
      K = (SHMEM_SIZE-16)/(2*dx*sizeof(_data_t));
      numBlocks = dim3(1,(dyNext+K-1)/K,dzNext);
      numThreads = dim3((T/K),K,1);
      mem = K*2*dx*sizeof(_data_t);

      cu_iwt3df_col <<< numBlocks,numThreads,mem >>>(dev_current_vx,dev_tempLx,dev_tempHx,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor1,dev_hir1,plan->filterLen);
      cuda_sync();

      //************WCY************
      // Update Pointers
      if (level==numLevels)
	dev_current_vy = dev_out_vy;
      // IWT Depths
      K = (SHMEM_SIZE-16)/(2*dz*sizeof(_data_t));
      numBlocks = dim3((dx+K-1)/K,dy,1);
      numThreads = dim3(K,1,(T/K));
      mem = K*2*dz*sizeof(_data_t);

      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempLxLy,dev_LxLyLz_df2,dev_LxLyHz_df2,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempHxLy,dev_HxLyLz_df2,dev_HxLyHz_df2,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempLxHy,dev_LxHyLz_df2,dev_LxHyHz_df2,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempHxHy,dev_HxHyLz_df2,dev_HxHyHz_df2,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,filterLen);
      cuda_sync();
      // IWT Rows
      K = (SHMEM_SIZE-16)/(2*dy*sizeof(_data_t));
      numBlocks = dim3((dx+K-1)/K,1,dzNext);
      numThreads = dim3(K,(T/K),1);
      mem = K*2*dy*sizeof(_data_t);

      cu_iwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempLx,dev_tempLxLy,dev_tempLxHy,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor1,dev_hir1,plan->filterLen);
      cu_iwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempHx,dev_tempHxLy,dev_tempHxHy,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor1,dev_hir1,plan->filterLen);
      cuda_sync();
      // IWT Columns
      K = (SHMEM_SIZE-16)/(2*dx*sizeof(_data_t));
      numBlocks = dim3(1,(dyNext+K-1)/K,dzNext);
      numThreads = dim3((T/K),K,1);
      mem = K*2*dx*sizeof(_data_t);

      cu_iwt3df_col <<< numBlocks,numThreads,mem >>>(dev_current_vy,dev_tempLx,dev_tempHx,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,plan->filterLen);
      cuda_sync();

      //************WCZ************
      // Update Pointers
      if (level==numLevels)
	dev_current_vz = dev_out_vz;
      // IWT Depths
      K = (SHMEM_SIZE-16)/(2*dz*sizeof(_data_t));
      numBlocks = dim3((dx+K-1)/K,dy,1);
      numThreads = dim3(K,1,(T/K));
      mem = K*2*dz*sizeof(_data_t);

      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempLxLy,dev_LxLyLz_n,dev_LxLyHz_n,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor1,dev_hir1,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempHxLy,dev_HxLyLz_n,dev_HxLyHz_n,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor1,dev_hir1,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempLxHy,dev_LxHyLz_n,dev_LxHyHz_n,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor1,dev_hir1,filterLen);
      cu_iwt3df_dep <<< numBlocks,numThreads,mem >>>(dev_tempHxHy,dev_HxHyLz_n,dev_HxHyHz_n,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor1,dev_hir1,filterLen);
      cuda_sync();
      // IWT Rows
      K = (SHMEM_SIZE-16)/(2*dy*sizeof(_data_t));
      numBlocks = dim3((dx+K-1)/K,1,dzNext);
      numThreads = dim3(K,(T/K),1);
      mem = K*2*dy*sizeof(_data_t);

      cu_iwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempLx,dev_tempLxLy,dev_tempLxHy,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,plan->filterLen);
      cu_iwt3df_row <<< numBlocks,numThreads,mem >>>(dev_tempHx,dev_tempHxLy,dev_tempHxHy,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,plan->filterLen);
      cuda_sync();
      // IWT Columns
      K = (SHMEM_SIZE-16)/(2*dx*sizeof(_data_t));
      numBlocks = dim3(1,(dyNext+K-1)/K,dzNext);
      numThreads = dim3((T/K),K,1);
      mem = K*2*dx*sizeof(_data_t);

      cu_iwt3df_col <<< numBlocks,numThreads,mem >>>(dev_current_vz,dev_tempLx,dev_tempHx,dx,dy,dz,dxNext,dyNext,dzNext,xOffset,yOffset,zOffset,dev_lor0,dev_hir0,plan->filterLen);
      cuda_sync();
      dev_HxLyLz_df1 += 7*blockSize;
      dev_HxLyLz_df2 += 7*blockSize;
      dev_HxLyLz_n += 7*blockSize;

    }
  cuda(Free( dev_filters ));
  cuda(Free( dev_temp1 ));
  cuda(Free( dev_temp2 ));
  
  circunshift_gpu(plan,out_vx);
  circunshift_gpu(plan,out_vy);
  circunshift_gpu(plan,out_vz);
}

int rand_lim(int limit) {

  int divisor = RAND_MAX/(limit+1);
  int retval;

  do { 
    retval = rand() / divisor;
  } while (retval > limit);

  return retval;
}

void dfwavelet_new_randshift_gpu (struct dfwavelet_plan_s* plan) {
  int i;
  i = rand();
  for(i = 0; i < plan->numdims; i++) {
    // Determine maximum shift value for this dimension
    int log2dim = 1;
    while( (1<<log2dim) < plan->imSize[i]) {
      log2dim++;
    }
    int maxShift = 1 << (log2dim-plan->numLevels);
    if (maxShift > 8) {
      maxShift = 8;
    }
    // Generate random shift value between 0 and maxShift
    plan->randShift[i] = rand_lim(maxShift);
  }
}

extern "C" void dfwavthresh3_gpu(struct dfwavelet_plan_s* plan,scalar_t dfthresh, scalar_t nthresh,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  data_t* dev_wcdf1,*dev_wcdf2,*dev_wcn;
  cuda(Malloc( (void**)&dev_wcdf1, plan->numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_wcdf2, plan->numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_wcn, plan->numCoeff*sizeof(_data_t) ));

  dffwt3_gpu(plan,dev_wcdf1,dev_wcdf2,dev_wcn,in_vx,in_vy,in_vz);
  dfsoftthresh_gpu(plan,dfthresh,nthresh,dev_wcdf1,dev_wcdf2,dev_wcn);
  dfiwt3_gpu(plan,out_vx,out_vy,out_vz,dev_wcdf1,dev_wcdf2,dev_wcn);

  cuda(Free( dev_wcdf1 ));
  cuda(Free( dev_wcdf2 ));
  cuda(Free( dev_wcn ));
}

extern "C" void dfwavthresh3_spin_gpu(struct dfwavelet_plan_s* plan,scalar_t dfthresh, scalar_t nthresh,int spins,int isRand,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  data_t *temp_vx,*temp_vy,*temp_vz;
  cuda(Malloc( (void**)&temp_vx, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_vy, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_vz, plan->numPixel*sizeof(_data_t) ));

  data_t *temp_out_vx,*temp_out_vy,*temp_out_vz;
  cuda(Malloc( (void**)&temp_out_vx, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_out_vy, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_out_vz, plan->numPixel*sizeof(_data_t) ));
  hipMemset(temp_out_vx,0,sizeof(data_t)*plan->numPixel);
  hipMemset(temp_out_vy,0,sizeof(data_t)*plan->numPixel);
  hipMemset(temp_out_vz,0,sizeof(data_t)*plan->numPixel);

  int T = 512;
  dim3 numThreads(T,1);
  dim3 numBlocks( (plan->numPixel+T-1)/T, 1);
  int s1,s2,s3;
  for (s1=0;s1<spins;s1++)
    for (s2=0;s2<spins;s2++)
      for (s3=0;s3<spins;s3++)
	{
	  if (isRand)
	    {
	      dfwavelet_new_randshift_gpu(plan);
	    } else 
	    {
	      plan->randShift[0] = s1;
	      plan->randShift[1] = s2;
	      plan->randShift[2] = s3;
	    }
	  dfwavthresh3_gpu(plan,dfthresh,nthresh,temp_vx,temp_vy,temp_vz,in_vx,in_vy,in_vz);
	  
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vx,temp_vx,plan->numPixel);
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vy,temp_vy,plan->numPixel);
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vz,temp_vz,plan->numPixel);
	}
  cuda(Memcpy(out_vx,temp_out_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));
  cuda(Memcpy(out_vy,temp_out_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));
  cuda(Memcpy(out_vz,temp_out_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));

  cu_mult<<<numBlocks,numThreads>>>(out_vx,1.0/((scalar_t) spins*spins*spins),plan->numPixel);
  cu_mult<<<numBlocks,numThreads>>>(out_vy,1.0/((scalar_t) spins*spins*spins),plan->numPixel);
  cu_mult<<<numBlocks,numThreads>>>(out_vz,1.0/((scalar_t) spins*spins*spins),plan->numPixel);

  cuda(Free( temp_vx ));
  cuda(Free( temp_vy ));
  cuda(Free( temp_vz ));
  cuda(Free( temp_out_vx ));
  cuda(Free( temp_out_vy ));
  cuda(Free( temp_out_vz ));
}

extern "C" void dfwavthresh3_SURE_gpu(struct dfwavelet_plan_s* plan,scalar_t sigma,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  data_t* dev_wcdf1,*dev_wcdf2,*dev_wcn;
  cuda(Malloc( (void**)&dev_wcdf1, plan->numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_wcdf2, plan->numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_wcn, plan->numCoeff*sizeof(_data_t) ));

  count_zeros_gpu(plan,in_vx,in_vy,in_vz);

  dffwt3_gpu(plan,dev_wcdf1,dev_wcdf2,dev_wcn,in_vx,in_vy,in_vz);
  dfSUREshrink_gpu(plan,sigma,dev_wcdf1,dev_wcdf2,dev_wcn);
  dfiwt3_gpu(plan,out_vx,out_vy,out_vz,dev_wcdf1,dev_wcdf2,dev_wcn);

  cuda(Free( dev_wcdf1 ));
  cuda(Free( dev_wcdf2 ));
  cuda(Free( dev_wcn ));
}

extern "C" void dfwavthresh3_SURE_spin_gpu(struct dfwavelet_plan_s* plan,scalar_t sigma,int spins,int isRand,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  data_t *temp_vx,*temp_vy,*temp_vz;
  cuda(Malloc( (void**)&temp_vx, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_vy, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_vz, plan->numPixel*sizeof(_data_t) ));

  data_t *temp_out_vx,*temp_out_vy,*temp_out_vz;
  cuda(Malloc( (void**)&temp_out_vx, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_out_vy, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_out_vz, plan->numPixel*sizeof(_data_t) ));
  hipMemset(temp_out_vx,0,sizeof(data_t)*plan->numPixel);
  hipMemset(temp_out_vy,0,sizeof(data_t)*plan->numPixel);
  hipMemset(temp_out_vz,0,sizeof(data_t)*plan->numPixel);

  count_zeros_gpu(plan,in_vx,in_vy,in_vz);

  int T = 512;
  dim3 numThreads(T,1);
  dim3 numBlocks( (plan->numPixel+T-1)/T, 1);
  int s1,s2,s3;
  for (s1=0;s1<spins;s1++)
    for (s2=0;s2<spins;s2++)
      for (s3=0;s3<spins;s3++)
	{
	  if (isRand)
	    {
	      dfwavelet_new_randshift_gpu(plan);
	    } else 
	    {
	      plan->randShift[0] = s1;
	      plan->randShift[1] = s2;
	      plan->randShift[2] = s3;
	    }
	  dfwavthresh3_SURE_gpu(plan,sigma,temp_vx,temp_vy,temp_vz,in_vx,in_vy,in_vz);
	  
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vx,temp_vx,plan->numPixel);
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vy,temp_vy,plan->numPixel);
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vz,temp_vz,plan->numPixel);
	}
  cuda(Memcpy(out_vx,temp_out_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));
  cuda(Memcpy(out_vy,temp_out_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));
  cuda(Memcpy(out_vz,temp_out_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));

  cu_mult<<<numBlocks,numThreads>>>(out_vx,1.0/((scalar_t) spins*spins*spins),plan->numPixel);
  cu_mult<<<numBlocks,numThreads>>>(out_vy,1.0/((scalar_t) spins*spins*spins),plan->numPixel);
  cu_mult<<<numBlocks,numThreads>>>(out_vz,1.0/((scalar_t) spins*spins*spins),plan->numPixel);

  cuda(Free( temp_vx ));
  cuda(Free( temp_vy ));
  cuda(Free( temp_vz ));
  cuda(Free( temp_out_vx ));
  cuda(Free( temp_out_vy ));
  cuda(Free( temp_out_vz ));
}

extern "C" void dfwavthresh3_SURE_MAD_gpu(struct dfwavelet_plan_s* plan,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  data_t* dev_wcdf1,*dev_wcdf2,*dev_wcn;
  cuda(Malloc( (void**)&dev_wcdf1, plan->numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_wcdf2, plan->numCoeff*sizeof(_data_t) ));
  cuda(Malloc( (void**)&dev_wcn, plan->numCoeff*sizeof(_data_t) ));

  count_zeros_gpu(plan,in_vx,in_vy,in_vz);

  dffwt3_gpu(plan,dev_wcdf1,dev_wcdf2,dev_wcn,in_vx,in_vy,in_vz);
  scalar_t sigma = getMADsigma_gpu(plan,dev_wcn);
  dfSUREshrink_gpu(plan,sigma,dev_wcdf1,dev_wcdf2,dev_wcn);
  dfiwt3_gpu(plan,out_vx,out_vy,out_vz,dev_wcdf1,dev_wcdf2,dev_wcn);

  cuda(Free( dev_wcdf1 ));
  cuda(Free( dev_wcdf2 ));
  cuda(Free( dev_wcn ));
}

extern "C" void dfwavthresh3_SURE_MAD_spin_gpu(struct dfwavelet_plan_s* plan,int spins,int isRand,data_t* out_vx,data_t* out_vy,data_t* out_vz,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{
  data_t *temp_vx,*temp_vy,*temp_vz;
  cuda(Malloc( (void**)&temp_vx, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_vy, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_vz, plan->numPixel*sizeof(_data_t) ));

  data_t *temp_out_vx,*temp_out_vy,*temp_out_vz;
  cuda(Malloc( (void**)&temp_out_vx, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_out_vy, plan->numPixel*sizeof(_data_t) ));
  cuda(Malloc( (void**)&temp_out_vz, plan->numPixel*sizeof(_data_t) ));
  hipMemset(temp_out_vx,0,sizeof(data_t)*plan->numPixel);
  hipMemset(temp_out_vy,0,sizeof(data_t)*plan->numPixel);
  hipMemset(temp_out_vz,0,sizeof(data_t)*plan->numPixel);

  count_zeros_gpu(plan,in_vx,in_vy,in_vz);

  int T = 512;
  dim3 numThreads(T,1);
  dim3 numBlocks( (plan->numPixel+T-1)/T, 1);
  int s1,s2,s3;
  for (s1=0;s1<spins;s1++)
    for (s2=0;s2<spins;s2++)
      for (s3=0;s3<spins;s3++)
	{
	  if (isRand)
	    {
	      dfwavelet_new_randshift_gpu(plan);
	    } else 
	    {
	      plan->randShift[0] = s1;
	      plan->randShift[1] = s2;
	      plan->randShift[2] = s3;
	    }
	  dfwavthresh3_SURE_MAD_gpu(plan,temp_vx,temp_vy,temp_vz,in_vx,in_vy,in_vz);
	  
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vx,temp_vx,plan->numPixel);
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vy,temp_vy,plan->numPixel);
	  cu_add<<<numBlocks,numThreads>>>(temp_out_vz,temp_vz,plan->numPixel);
	}
  cuda(Memcpy(out_vx,temp_out_vx, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));
  cuda(Memcpy(out_vy,temp_out_vy, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));
  cuda(Memcpy(out_vz,temp_out_vz, plan->numPixel*sizeof(data_t), hipMemcpyDeviceToDevice));

  cu_mult<<<numBlocks,numThreads>>>(out_vx,1.0/((scalar_t) spins*spins*spins),plan->numPixel);
  cu_mult<<<numBlocks,numThreads>>>(out_vy,1.0/((scalar_t) spins*spins*spins),plan->numPixel);
  cu_mult<<<numBlocks,numThreads>>>(out_vz,1.0/((scalar_t) spins*spins*spins),plan->numPixel);

  cuda(Free( temp_vx ));
  cuda(Free( temp_vy ));
  cuda(Free( temp_vz ));
  cuda(Free( temp_out_vx ));
  cuda(Free( temp_out_vy ));
  cuda(Free( temp_out_vz ));
}

scalar_t getMADsigma_gpu(struct dfwavelet_plan_s* plan, data_t* wcn)
{
  data_t* subband = wcn + plan->waveSizes[0]*plan->waveSizes[1]*plan->waveSizes[2];
  int l;
  for (l = 1; l <= plan->numLevels; ++l){
    subband += 7*plan->waveSizes[0 + 3*l]*plan->waveSizes[1 + 3*l]*plan->waveSizes[2 + 3*l];
  }

  int dx = plan->waveSizes[0 + 3*plan->numLevels];
  int dy = plan->waveSizes[1 + 3*plan->numLevels];
  int dz = plan->waveSizes[2 + 3*plan->numLevels];
  int blockSize = dx*dy*dz;
  int numZeros = (plan->percentZero*blockSize);
  int num = blockSize-numZeros;
  subband = subband - blockSize;

  int const T = 512;
  dim3 numBlocks = dim3((blockSize+T-1)/T,1);
  dim3 numThreads = dim3(T,1);

  scalar_t *subband2;
  cuda(Malloc( (void**)&subband2, blockSize*sizeof(scalar_t) ));
  cu_subband2<<<numBlocks,numThreads>>>(subband2,subband,blockSize);

  bitonic_sort(subband2,blockSize);
  scalar_t sigma;
  cuda(Memcpy(&sigma, subband2 + numZeros + num/2 , sizeof(scalar_t), hipMemcpyDeviceToHost));
  // Scale by 1/noiseAMP for that subband
  sigma = 1.4826*sqrt(sigma)/plan->noiseAmp[20];
  cuda(Free(subband2));

  return sigma;
}


extern "C" void dfsoftthresh_gpu(struct dfwavelet_plan_s* plan,scalar_t dfthresh, scalar_t nthresh, data_t* wcdf1,data_t* wcdf2,data_t* wcn)
{
	data_t* HxLyLz1 = wcdf1 + plan->waveSizes[0]*plan->waveSizes[1]*plan->waveSizes[2];
	data_t* HxLyLz2 = wcdf2 + plan->waveSizes[0]*plan->waveSizes[1]*plan->waveSizes[2];
	data_t* HxLyLz3 = wcn + plan->waveSizes[0]*plan->waveSizes[1]*plan->waveSizes[2];
	int l;
	for (l = 1; l <= plan->numLevels; ++l){
		HxLyLz1 += 7*plan->waveSizes[0 + 3*l]*plan->waveSizes[1 + 3*l]*plan->waveSizes[2 + 3*l];
		HxLyLz2 += 7*plan->waveSizes[0 + 3*l]*plan->waveSizes[1 + 3*l]*plan->waveSizes[2 + 3*l];
		HxLyLz3 += 7*plan->waveSizes[0 + 3*l]*plan->waveSizes[1 + 3*l]*plan->waveSizes[2 + 3*l];
	}
	int dxNext = plan->waveSizes[0 + 3*plan->numLevels];
	int dyNext = plan->waveSizes[1 + 3*plan->numLevels];
	int dzNext = plan->waveSizes[2 + 3*plan->numLevels];
	int blockSize = dxNext*dyNext*dzNext;
	int naInd = 0;

	for (l = plan->numLevels; l >= 1; --l)
	{
		dxNext = plan->waveSizes[0 + 3*l];
		dyNext = plan->waveSizes[1 + 3*l];
		dzNext = plan->waveSizes[2 + 3*l];
		blockSize = dxNext*dyNext*dzNext;

		HxLyLz1 = HxLyLz1 - 7*blockSize;
		HxLyLz2 = HxLyLz2 - 7*blockSize;
		HxLyLz3 = HxLyLz3 - 7*blockSize;

		int bandInd;
		for (bandInd=0; bandInd<7*3;bandInd++)
		{
			data_t *subband;
			scalar_t lambda;
			if (bandInd<7)
			{
				subband = HxLyLz1 + bandInd*blockSize;
				lambda = dfthresh * plan->noiseAmp[naInd];
			} else if (bandInd<14)
			{
				subband = HxLyLz2 + (bandInd-7)*blockSize;
				lambda = dfthresh * plan->noiseAmp[naInd];
			} else
			{
				subband = HxLyLz3 + (bandInd-14)*blockSize;
				lambda = nthresh * plan->noiseAmp[naInd];
			}

			// SoftThresh
			softthresh_gpu(plan, blockSize, lambda,subband);
			naInd++;
	  
		} 
	}
}

void dfSUREshrink_gpu(struct dfwavelet_plan_s* plan,scalar_t sigma,data_t* dev_wcdf1,data_t* dev_wcdf2,data_t* dev_wcn)
{
  scalar_t percentZero = plan->percentZero;
  data_t* HxLyLz1 = dev_wcdf1 + plan->waveSizes[0]*plan->waveSizes[1]*plan->waveSizes[2];
  data_t* HxLyLz2 = dev_wcdf2 + plan->waveSizes[0]*plan->waveSizes[1]*plan->waveSizes[2];
  data_t* HxLyLz3 = dev_wcn + plan->waveSizes[0]*plan->waveSizes[1]*plan->waveSizes[2];
  int l;
  for (l = 1; l <= plan->numLevels; ++l){
    HxLyLz1 += 7*plan->waveSizes[0 + 3*l]*plan->waveSizes[1 + 3*l]*plan->waveSizes[2 + 3*l];
    HxLyLz2 += 7*plan->waveSizes[0 + 3*l]*plan->waveSizes[1 + 3*l]*plan->waveSizes[2 + 3*l];
    HxLyLz3 += 7*plan->waveSizes[0 + 3*l]*plan->waveSizes[1 + 3*l]*plan->waveSizes[2 + 3*l];
  }
  int dxNext = plan->waveSizes[0 + 3*plan->numLevels];
  int dyNext = plan->waveSizes[1 + 3*plan->numLevels];
  int dzNext = plan->waveSizes[2 + 3*plan->numLevels];
  int blockSize = dxNext*dyNext*dzNext;
  scalar_t *subband2, *cum_sum;
  cuda(Malloc( (void**)&subband2, blockSize*sizeof(scalar_t) ));
  cuda(Malloc( (void**)&cum_sum, blockSize*sizeof(scalar_t) ));
  int naInd = 0;

  for (l = plan->numLevels; l >= 1; --l)
    {
      dxNext = plan->waveSizes[0 + 3*l];
      dyNext = plan->waveSizes[1 + 3*l];
      dzNext = plan->waveSizes[2 + 3*l];
      blockSize = dxNext*dyNext*dzNext;

      HxLyLz1 = HxLyLz1 - 7*blockSize;
      HxLyLz2 = HxLyLz2 - 7*blockSize;
      HxLyLz3 = HxLyLz3 - 7*blockSize;

      int bandInd;

      for (bandInd=0; bandInd<7*3;bandInd++)
	{
	  data_t *subband;
	  if (bandInd<7)
	    {
	      subband = HxLyLz1 + bandInd*blockSize;
	    } else if (bandInd<14)
	    {
	      subband = HxLyLz2 + (bandInd-7)*blockSize;
	    } else
	    {
	      subband = HxLyLz3 + (bandInd-14)*blockSize;
	    }
	  // Scale sigma by noise amplifiction in each subband
	  scalar_t sigma_band = sigma*plan->noiseAmp[naInd];
	  scalar_t sigma2 = sigma_band*sigma_band;
	  naInd++;

	  // Sort, Get cum_sum
	  scalar_t ener;
	  int const T = 512;
	  dim3 numBlocks = dim3((blockSize+T-1)/T,1);
	  dim3 numThreads = dim3(T,1);
	  cu_subband2<<<numBlocks,numThreads>>>(subband2,subband,blockSize);
	  bitonic_sort(subband2,blockSize);
	  prefix_sum(&ener,cum_sum,subband2,blockSize);

	  // N = blockSize*(1-%zero)
	  int numZeros = (percentZero*blockSize);
	  int num = blockSize-numZeros;

	  scalar_t thresh;
	  // Choose visu or sure
	  /*scalar_t density = (ener/num/sigma2-1);
	  scalar_t critical = (scalar_t) pow(log2 ((double) num),1.5)/sqrt(num);

	  if (density < critical)
	    {
	      thresh = sqrt(2*sigma2*log(num)); //visuShrink
	      } else*/
	    {
	      // Get SURE threshold
	      thresh = get_SURE_thresh(subband2+numZeros,cum_sum+numZeros,sigma2,num);
	    }
	    //printf("Sigma = %f, Sigma2 = %f, Thresh = %f, VisuThresh = %f, Ener = %f, percentZero = %f\n",sigma,sigma2,thresh,sqrt(sigma2*2*log(num)),ener,percentZero);

	  // SoftThresh
	  numBlocks = dim3((blockSize+T-1)/T,1);
	  numThreads = dim3(T,1);
	  cu_soft_thresh <<< numBlocks,numThreads>>> (subband,thresh,blockSize);
	} 
    }
  cuda(Free(subband2));
  cuda(Free(cum_sum));
}

__global__ void cu_subband2 (scalar_t* subband2, data_t* subband,int length)
{
  int const i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i>length)
    return;

  subband2[i] = abs(subband[i])*abs(subband[i]);
}

__global__ void cu_calc_SURE_risk (scalar_t* risk, scalar_t* subband2, scalar_t* cum_sum,scalar_t sigma2,int length)
{
  int const i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i>length)
    return;

  risk[i] = -( 2*sigma2*(i+1))+cum_sum[i]+subband2[i]*(length-i-1);
}

__global__ void cu_find_min_risk (scalar_t* out_risk,scalar_t* out_subband2,scalar_t* in_risk,scalar_t* in_subband2,int length)
{
  extern __shared__ scalar_t temp[];
  int const i = threadIdx.x + blockDim.x*blockIdx.x;
  int thid = threadIdx.x;
  int blid = blockIdx.x;
  scalar_t* temp_risk = temp;
  scalar_t* temp_subband2 = temp+blockDim.x;
  if (i<length)
    {
      temp_risk[thid] = in_risk[i]; 
      temp_subband2[thid] = in_subband2[i];
    } else
    {
      temp_risk[thid] = 3e38;
      temp_subband2[thid] = 0;
    }
  
  for (int jump = blockDim.x>>1; jump > 0; jump >>= 1) 
    {   
      __syncthreads();  
      int ai = thid;  
      int bi = thid+jump; 

      if (ai < jump)
	if (temp_risk[bi]<temp_risk[ai])
	  {
	    temp_risk[ai] = temp_risk[bi]; 
	    temp_subband2[ai] = temp_subband2[bi];
	  }
    }
  __syncthreads();  
  
  if (thid==0) 
    out_risk[blid] = temp_risk[0]; out_subband2[blid] = temp_subband2[0];
}

scalar_t get_SURE_thresh (scalar_t* subband2, scalar_t* cum_sum,scalar_t sigma2,int length)
{
  scalar_t* risk;
  cuda(Malloc( (void**)&risk, (length)*sizeof(scalar_t) ));

  int const T = 512;
  dim3 numThreads(T,1);
  dim3 numBlocks( (length+T-1)/T, 1);
  // Map
  cu_calc_SURE_risk <<< numBlocks, numThreads >>>(risk,subband2,cum_sum,sigma2,length);
  cuda_sync();
  // Reduce
  scalar_t* in_risk = risk;
  scalar_t* in_subband2 = subband2;
  scalar_t* temp;
  cuda(Malloc( (void**)&temp, length*2*sizeof(scalar_t) ));
  scalar_t* out_risk = temp;
  scalar_t* out_subband2 = temp+length;

  int mem = T*sizeof(_data_t)*2;
  scalar_t* result_risk = out_risk;
  scalar_t* result_subband2 = out_subband2;
  int l = length;
  do {
      numBlocks = dim3( (l+T-1)/T, 1);
      cu_find_min_risk <<< numBlocks,numThreads,mem>>> (out_risk,out_subband2,in_risk,in_subband2,l);
      cuda_sync();

      result_risk = out_risk;
      result_subband2 = out_subband2;
      out_risk = in_risk;
      out_subband2 = in_subband2;
      in_risk = result_risk;
      in_subband2 = result_subband2;
      l = numBlocks.x;
  } while (numBlocks.x > 1);
  scalar_t thresh;
  cuda(Memcpy(&thresh, result_subband2, sizeof(scalar_t), hipMemcpyDeviceToHost));
  
  thresh = sqrtf(thresh);

  if (thresh>sqrtf(2*sigma2*log(length)))
    thresh = sqrtf(2*sigma2*log(length));
  
  cuda(Free(risk));
  cuda(Free(temp));
  return thresh;
}

__global__ void cu_is_zeros (scalar_t* isZeros,data_t* in_vx,data_t* in_vy,data_t* in_vz,int length)
{
  int const i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i<length)
    isZeros[i] = ((in_vx[i]==0.)&&(in_vy[i]==0.)&&(in_vz[i]==0.));
}

__global__ void cu_sumAll (scalar_t* out,scalar_t* in, int length)
{
  extern __shared__ scalar_t temp[];
  int const i = threadIdx.x + blockDim.x*blockIdx.x;
  int thid = threadIdx.x;
  int blid = blockIdx.x;
  if (i<length)
    temp[thid] = in[i];
  else
    temp[thid] = 0;

  for (int jump = blockDim.x>>1; jump > 0; jump >>= 1) 
    {   
      __syncthreads();  
      int ai = thid;  
      int bi = thid+jump; 

      if (ai < jump)
	temp[ai]+=temp[bi];
    }
  __syncthreads();  
  
  if (thid==0) 
    out[blid] = temp[0];
}

void count_zeros_gpu (struct dfwavelet_plan_s* plan,data_t* in_vx,data_t* in_vy,data_t* in_vz)
{

  if (plan->percentZero==-1)
    {
      int length = plan->numPixel;
      scalar_t* isZeros;
      cuda(Malloc( (void**)&isZeros, length*sizeof(scalar_t) ));

      int const T = 512;
      dim3 numThreads(T,1);
      dim3 numBlocks( (length+T-1)/T, 1);
      // Map
      cu_is_zeros <<< numBlocks, numThreads >>>(isZeros,in_vx,in_vy,in_vz,length);
      cuda_sync();
      // Reduce
      scalar_t* in_numZeros = isZeros;
      scalar_t* temp;
      cuda(Malloc( (void**)&temp, length*sizeof(scalar_t) ));
      scalar_t* out_numZeros = temp;

      int mem = T*sizeof(_data_t);
      scalar_t* result = out_numZeros;
      int l = length;
      do {
	numBlocks = dim3( (l+T-1)/T, 1);
	cu_sumAll <<< numBlocks,numThreads,mem>>> (out_numZeros,in_numZeros,l);
	cuda_sync();
	result = out_numZeros;
	out_numZeros = in_numZeros;
	in_numZeros = result;
	l = numBlocks.x;
      } while (numBlocks.x > 1);
      cuda(Memcpy(&plan->percentZero, result, sizeof(scalar_t), hipMemcpyDeviceToHost));
      plan->percentZero = plan->percentZero/length;
      cuda(Free(isZeros));
      cuda(Free(temp));
    }
}

__global__ void cu_bitonic_sort_step(scalar_t *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        scalar_t temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        scalar_t temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

/**
 * Bitonic sort using CUDA, accepts only nonnegative numbers
 */
void bitonic_sort(scalar_t *dev_values,int length)
{
  int length2 = 1;
  int l = length;
  while (l>>=1)
    length2<<=1;
  if (length2!=length)
    length2<<=1;

  scalar_t *dev_temp_values;
  hipMalloc((void**) &dev_temp_values, length2*sizeof(scalar_t));
  hipMemset(dev_temp_values+length,0,(length2-length)*sizeof(scalar_t));
  hipMemcpy(dev_temp_values, dev_values, length*sizeof(scalar_t), hipMemcpyDeviceToDevice);
  
  int const T = (length2>512)? 512 : length2;
  dim3 numBlocks(length2/T,1);    /* Number of blocks   */
  dim3 numThreads(T,1);  /* Number of threads  */

  int j, k;
  for (k = 2; k <= length2; k <<= 1) {
    for (j=k>>1; j>0; j=j>>1) {
      cu_bitonic_sort_step<<<numBlocks, numThreads>>>(dev_temp_values, j, k);
    }
  }
  cuda(Memcpy(dev_values, dev_temp_values+(length2-length), length*sizeof(scalar_t), hipMemcpyDeviceToDevice));
  hipFree(dev_temp_values);
}


__global__ void cu_prescan(scalar_t* out,scalar_t *in, int n)  
{   
  extern __shared__ scalar_t temp[];  // allocated on invocation  
  int thid = threadIdx.x;  
  int offset = 1;  
 	
  temp[2*thid] = in[2*thid]; // load input into shared memory  
  temp[2*thid+1] = in[2*thid+1]; 

  for (int d = n>>1; d > 0; d >>= 1)                    // build sum in place up the tree  
    {   
      __syncthreads();  
      if (thid < d)  
	{  
	  int ai = offset*(2*thid+1)-1;  
	  int bi = offset*(2*thid+2)-1;  
 	
	  temp[bi] += temp[ai];  
	}  
      offset *= 2;  
    }

  if (thid == 0) { temp[n - 1] = 0; } // clear the last element   

  for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
    {  
      offset >>= 1;  
      __syncthreads();  
      if (thid < d)                       
	{  
     
	  int ai = offset*(2*thid+1)-1;  
	  int bi = offset*(2*thid+2)-1;  
 	
	  scalar_t t = temp[ai];  
	  temp[ai] = temp[bi];  
	  temp[bi] += t;   
	}  
    }  

  __syncthreads();  
  out[2*thid] = temp[2*thid];
  out[2*thid+1] = temp[2*thid+1];
}  

__global__ void cu_add_cum(scalar_t *out,scalar_t* in)
{
  int thid = threadIdx.x;
  out[thid] += in[-1];
  out[thid] += out[-1];
}

void prefix_sum(scalar_t* ener,scalar_t* out_values, scalar_t *in_values,int length)
{
  int length2 = 1;
  int l = length;
  while (l>>=1)
    length2<<=1;
  if (length2!=length)
    length2<<=1;
  scalar_t *dev_temp_values_in,*dev_temp_values_out;
  hipMalloc((void**) &dev_temp_values_in, length2*sizeof(scalar_t));
  hipMalloc((void**) &dev_temp_values_out, length2*sizeof(scalar_t));
  hipMemset(dev_temp_values_in+length,0,(length2-length)*sizeof(scalar_t));
  hipMemcpy(dev_temp_values_in, in_values, length*sizeof(scalar_t), hipMemcpyDeviceToDevice);
  int T = (length2>512)? 512 : length2;
  dim3 numBlocks(length2/T,1);    /* Number of blocks   */
  dim3 numThreads(T/2,1);  /* Number of threads  */
  int mem = T*sizeof(_data_t);

  int i;
  for (i=0; i < length2/T; i++)
    {
      cu_prescan<<<numBlocks, numThreads, mem>>>(dev_temp_values_out+i*T,dev_temp_values_in+i*T,T);
      cuda_sync();
    }
  
  for (i = 1; i< numBlocks.x; i++)
    {
      cu_add_cum<<< 1, T >>> (dev_temp_values_out+i*T,dev_temp_values_in+i*T);
      cuda_sync();
    }
  
  hipMemcpy(out_values, dev_temp_values_out, length*sizeof(scalar_t), hipMemcpyDeviceToDevice);
  hipFree(dev_temp_values_in);
  hipFree(dev_temp_values_out);

  // Get Energy
  scalar_t ener_1;
  hipMemcpy(&ener_1, out_values+length-1, sizeof(scalar_t), hipMemcpyDeviceToHost);
  hipMemcpy(ener, in_values+length-1, sizeof(scalar_t), hipMemcpyDeviceToHost);
  ener[0] += ener_1;
}



/********** Aux functions **********/
extern "C" void softthresh_gpu (struct dfwavelet_plan_s* plan,int length, scalar_t thresh,data_t* coeff_c)
{
  assert(plan->use_gpu==1||plan->use_gpu==2);

  _data_t* dev_coeff;
  dev_coeff = (_data_t*) coeff_c;
  int numMax;
  int const T = 512;
  dim3 numBlocks, numThreads;
  numMax = length;
  numBlocks = dim3((numMax+T-1)/T,1,1);
  numThreads = dim3(T,1,1);
  cu_soft_thresh <<< numBlocks,numThreads>>> (dev_coeff,thresh,numMax);
}

extern "C" void circshift_gpu(struct dfwavelet_plan_s* plan, data_t* data_c) {
  // Return if no shifts
  int zeroShift = 1;
  int i;
  for (i = 0; i< plan->numdims; i++)
    {
      zeroShift &= (plan->randShift[i]==0);
    }
  if(zeroShift) {
    return;
  }
  _data_t* data = (_data_t*) data_c;
  // Copy data
  _data_t* dataCopy;
  cuda(Malloc((void**)&dataCopy, plan->numPixel*sizeof(_data_t)));
  cuda(Memcpy(dataCopy, data, plan->numPixel*sizeof(_data_t), hipMemcpyDeviceToDevice));
  int T = 512;
  if (plan->numdims==2)
    {
      int dx,dy,r0,r1;
      dx = plan->imSize[0];
      dy = plan->imSize[1];
      r0 = plan->randShift[0];
      r1 = plan->randShift[1];
      cu_circshift <<< (plan->numPixel+T-1)/T, T>>>(data,dataCopy,dx,dy,1,r0,r1,0);
    } else if (plan->numdims==3)
    {
      int dx,dy,dz,r0,r1,r2;
      dx = plan->imSize[0];
      dy = plan->imSize[1];
      dz = plan->imSize[2];
      r0 = plan->randShift[0];
      r1 = plan->randShift[1];
      r2 = plan->randShift[2];
      cu_circshift <<< (plan->numPixel+T-1)/T, T>>>(data,dataCopy,dx,dy,dz,r0,r1,r2);
    }
  cuda(Free(dataCopy));
}

extern "C" void circunshift_gpu(struct dfwavelet_plan_s* plan, data_t* data_c) {
  // Return if no shifts
  int zeroShift = 1;
  int i;
  for (i = 0; i< plan->numdims; i++)
    {
      zeroShift &= (plan->randShift[i]==0);
    }
  if(zeroShift) {
    return;
  }
  _data_t* data = (_data_t*) data_c;
  // Copy data
  _data_t* dataCopy;
  cuda(Malloc((void**)&dataCopy, plan->numPixel*sizeof(_data_t)));
  cuda(Memcpy(dataCopy, data, plan->numPixel*sizeof(_data_t), hipMemcpyDeviceToDevice));
  int T = 512;
  if (plan->numdims==2)
    {
      int dx,dy,r0,r1;
      dx = plan->imSize[0];
      dy = plan->imSize[1];
      r0 = plan->randShift[0];
      r1 = plan->randShift[1];
      cu_circunshift <<< (plan->numPixel+T-1)/T, T>>>(data,dataCopy,dx,dy,1,r0,r1,0);
    } else if (plan->numdims==3)
    {
      int dx,dy,dz,r0,r1,r2;
      dx = plan->imSize[0];
      dy = plan->imSize[1];
      dz = plan->imSize[2];
      r0 = plan->randShift[0];
      r1 = plan->randShift[1];
      r2 = plan->randShift[2];
      cu_circunshift <<< (plan->numPixel+T-1)/T, T>>>(data,dataCopy,dx,dy,dz,r0,r1,r2);
    }
  cuda(Free(dataCopy));
}

// ############################################################################
// CUDA function of fwt column convolution
// Loads data to scratchpad (shared memory) and convolve w/ low pass and high pass
// Output: Lx, Hx
// Input:  in, dx, dy, dz, dxNext, lod, hid, filterLen
// ############################################################################
extern "C" __global__ void cu_fwt3df_col(_data_t *Lx,_data_t *Hx,_data_t *in,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,scalar_t *lod,scalar_t *hid,int filterLen)
{
  extern __shared__ _data_t cols [];
  int ti = threadIdx.x;
  int tj = threadIdx.y;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;

  if (j>=dy) {
    return;
  }

  // Load Input to Temp Array
  for (int i = ti; i < dx; i += blockDim.x){
    cols[i + tj*dx] = in[i + j*dx + k*dx*dy];
  }
  __syncthreads();
  // Low-Pass and High-Pass Downsample
  int ind, lessThan, greaThan;
  for (int i = ti; i < dxNext; i += blockDim.x){
    _data_t y = cols[0]-cols[0];
    _data_t z = cols[0]-cols[0];
#pragma unroll
    for (int f = 0; f < filterLen; f++){
      ind = 2*i+1 - (filterLen-1)+f;

      lessThan = (int) (ind<0);
      greaThan = (int) (ind>=dx);
      ind = -1*lessThan+ind*(-2*lessThan+1);
      ind = (2*dx-1)*greaThan+ind*(-2*greaThan+1);

      y += cols[ind + tj*dx] * lod[filterLen-1-f];
      z += cols[ind + tj*dx] * hid[filterLen-1-f];
    }
    Lx[i + j*dxNext + k*dxNext*dy] = y;
    Hx[i + j*dxNext + k*dxNext*dy] = z;
  }
}

// ############################################################################
// CUDA function of fwt row convolution. Assumes fwt_col() has already been called
// Loads data to scratchpad (shared memory) and convolve w/ low pass and high pass
// Output: LxLy, LxHy / HxLy, HxHy
// Input:  Lx/Hx, dx, dy, dxNext, dyNext, lod, hid, filterLen
// ############################################################################
extern "C" __global__ void cu_fwt3df_row(_data_t *Ly,_data_t *Hy,_data_t *in,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,scalar_t *lod,scalar_t *hid,int filterLen)
{
  extern __shared__ _data_t rows [];
  int const K = blockDim.x;
  int ti = threadIdx.x;
  int tj = threadIdx.y;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int k = blockIdx.z*blockDim.z+threadIdx.z;

  if (i>=dxNext)
    {
      return;
    }

  for (int j = tj; j < dy; j += blockDim.y){
    rows[ti + j*K] = in[i + j*dxNext + k*dxNext*dy];
  }
  __syncthreads();

  // Low-Pass and High Pass Downsample
  int ind, lessThan, greaThan;
  for (int j = tj; j < dyNext; j += blockDim.y){
    _data_t y = rows[0]-rows[0];
    _data_t z = rows[0]-rows[0];
#pragma unroll
    for (int f = 0; f < filterLen; f++){
      ind = 2*j+1 - (filterLen-1)+f;
      lessThan = (int) (ind<0);
      greaThan = (int) (ind>=dy);
      ind = -1*lessThan+ind*(-2*lessThan+1);
      ind = (2*dy-1)*greaThan+ind*(-2*greaThan+1);
      y += rows[ti + ind*K] * lod[filterLen-1-f];
      z += rows[ti + ind*K] * hid[filterLen-1-f];
    }
    Ly[i + j*dxNext + k*dxNext*dyNext] = y;
    Hy[i + j*dxNext + k*dxNext*dyNext] = z;
  }
}

// ############################################################################
// CUDA function of fwt depth convolution. Assumes fwt_row() has already been called
// Loads data to scratchpad (shared memory) and convolve w/ low pass and high pass
// Output: LxLy, LxHy / HxLy, HxHy
// Input:  Lx/Hx, dx, dy, dxNext, dyNext, lod, hid, filterLen
// ############################################################################
extern "C" __global__ void cu_fwt3df_dep(_data_t *Lz,_data_t *Hz,_data_t *in,int dx,int dy,int dz,int dxNext,int dyNext,int dzNext,scalar_t *lod,scalar_t *hid,int filterLen)
{
  extern __shared__ _data_t deps [];
  int const K = blockDim.x;
  int ti = threadIdx.x;
  int tk = threadIdx.z;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;

  if (i>=dxNext)
    {
      return;
    }

  for (int k = tk; k < dz; k += blockDim.z){
    deps[ti + k*K] = in[i + j*dxNext + k*dxNext*dyNext];
  }
  __syncthreads();

  // Low-Pass and High Pass Downsample
  int ind, lessThan, greaThan;
  for (int k = tk; k < dzNext; k += blockDim.z){
    _data_t y = deps[0]-deps[0];
    _data_t z = deps[0]-deps[0];
#pragma unroll
    for (int f = 0; f < filterLen; f++){
      ind = 2*k+1 - (filterLen-1)+f;
      lessThan = (int) (ind<0);
      greaThan = (int) (ind>=dz);
      ind = -1*lessThan+ind*(-2*lessThan+1);
      ind = (2*dz-1)*greaThan+ind*(-2*greaThan+1);
      y += deps[ti + ind*K] * lod[filterLen-1-f];
      z += deps[ti + ind*K] * hid[filterLen-1-f];
    }
    Lz[i + j*dxNext + k*dxNext*dyNext] = y;
    Hz[i + j*dxNext + k*dxNext*dyNext] = z;
  }
}

extern "C" __global__ void cu_fwt3df_LC1(_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dxNext, int dyNext, int dzNext)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t x,y,z,xGreatZero,yGreatZero,zGreatZero;
  if ((i>=dxNext)||(j>=dyNext)||(k>=dzNext))
    {
      return;
    }

  //HLL
  x = HxLyLz_df1[i+j*dxNext+k*dxNext*dyNext];
  y = HxLyLz_df2[i+j*dxNext+k*dxNext*dyNext];
  z = HxLyLz_n[i+j*dxNext+k*dxNext*dyNext];
  HxLyLz_df1[i+j*dxNext+k*dxNext*dyNext] = y;
  HxLyLz_df2[i+j*dxNext+k*dxNext*dyNext] = z;
  yGreatZero = j>0;
  zGreatZero = k>0;
  HxLyLz_n[i+j*dxNext+k*dxNext*dyNext] = x + yGreatZero*0.25*y + zGreatZero*0.25*z;

  //LHL
  x = LxHyLz_df1[i+j*dxNext+k*dxNext*dyNext];
  y = LxHyLz_df2[i+j*dxNext+k*dxNext*dyNext];
  z = LxHyLz_n[i+j*dxNext+k*dxNext*dyNext];
  LxHyLz_df2[i+j*dxNext+k*dxNext*dyNext] = z;
  xGreatZero = i>0;
  zGreatZero = k>0;
  LxHyLz_n[i+j*dxNext+k*dxNext*dyNext] = y + xGreatZero*0.25*x + zGreatZero*0.25*z;
      
  //LLH
  x = LxLyHz_df1[i+j*dxNext+k*dxNext*dyNext];
  y = LxLyHz_df2[i+j*dxNext+k*dxNext*dyNext];
  z = LxLyHz_n[i+j*dxNext+k*dxNext*dyNext];
  LxLyHz_df1[i+j*dxNext+k*dxNext*dyNext] = y;
  LxLyHz_df2[i+j*dxNext+k*dxNext*dyNext] = x;
  yGreatZero = j>0;
  xGreatZero = i>0;
  LxLyHz_n[i+j*dxNext+k*dxNext*dyNext] = z + yGreatZero*0.25*y + xGreatZero*0.25*x;
}
extern "C" __global__ void cu_fwt3df_LC1_diff(_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dxNext, int dyNext, int dzNext)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t x,y,z;
  if ((i>=dxNext)||(j>=dyNext)||(k>=dzNext))
    {
      return;
    }

  //HLL
  if (j>0)
    y = HxLyLz_df1[i+(j-1)*dxNext+k*dxNext*dyNext];
  else
    y = 0;
  if (k>0)
    z = HxLyLz_df2[i+j*dxNext+(k-1)*dxNext*dyNext];
  else
    z = 0;
  HxLyLz_n[i+j*dxNext+k*dxNext*dyNext] += -0.25*y - 0.25*z;

  //LHL
  if (i>0)
    x = LxHyLz_df1[(i-1)+j*dxNext+k*dxNext*dyNext];
  else
    x = 0;
  if (k>0)
    z = LxHyLz_df2[i+j*dxNext+(k-1)*dxNext*dyNext];
  else
    z = 0;
  LxHyLz_n[i+j*dxNext+k*dxNext*dyNext] += -0.25*x - 0.25*z;

  //LLH
  if (j>0)
    y = LxLyHz_df1[i+(j-1)*dxNext+k*dxNext*dyNext];
  else
    y = 0;
  if (i>0)
    x = LxLyHz_df2[(i-1)+j*dxNext+k*dxNext*dyNext];
  else
    x = 0;
  LxLyHz_n[i+j*dxNext+k*dxNext*dyNext] += -0.25*y - 0.25*x;
}
extern "C" __global__ void cu_fwt3df_LC2(_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dxNext, int dyNext, int dzNext)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t x,y,z,xGreatZero,yGreatZero,zGreatZero;
  if ((i>=dxNext)||(j>=dyNext)||(k>=dzNext))
    {
      return;
    }

  //HHL
  x = HxHyLz_df1[i+j*dxNext+k*dxNext*dyNext];
  y = HxHyLz_df2[i+j*dxNext+k*dxNext*dyNext];
  z = HxHyLz_n[i+j*dxNext+k*dxNext*dyNext];
  HxHyLz_df1[i+j*dxNext+k*dxNext*dyNext] = 0.5*(x-y);
  HxHyLz_df2[i+j*dxNext+k*dxNext*dyNext] = z;
  zGreatZero = k>0;
  HxHyLz_n[i+j*dxNext+k*dxNext*dyNext] = 0.5*(x+y) + zGreatZero*0.125*z;

  //HLH
  x = HxLyHz_df1[i+j*dxNext+k*dxNext*dyNext];
  y = HxLyHz_df2[i+j*dxNext+k*dxNext*dyNext];
  z = HxLyHz_n[i+j*dxNext+k*dxNext*dyNext];
  HxLyHz_df1[i+j*dxNext+k*dxNext*dyNext] = 0.5*(z-x);
  HxLyHz_df2[i+j*dxNext+k*dxNext*dyNext] = y;
  yGreatZero = j>0;
  HxLyHz_n[i+j*dxNext+k*dxNext*dyNext] = 0.5*(z+x) + yGreatZero*0.125*y;
      
  //LHH
  x = LxHyHz_df1[i+j*dxNext+k*dxNext*dyNext];
  y = LxHyHz_df2[i+j*dxNext+k*dxNext*dyNext];
  z = LxHyHz_n[i+j*dxNext+k*dxNext*dyNext];
  LxHyHz_df1[i+j*dxNext+k*dxNext*dyNext] = 0.5*(y-z);
  LxHyHz_df2[i+j*dxNext+k*dxNext*dyNext] = x;
  xGreatZero = i>0;
  LxHyHz_n[i+j*dxNext+k*dxNext*dyNext] = 0.5*(y+z) + xGreatZero*0.125*x;
}

extern "C" __global__ void cu_fwt3df_LC2_diff(_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dxNext, int dyNext, int dzNext)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t x,y,z;
  if ((i>=dxNext)||(j>=dyNext)||(k>=dzNext))
    {
      return;
    }

  //HHL
  if (k>0)
    z = HxHyLz_df2[i+j*dxNext+(k-1)*dxNext*dyNext];
  else 
    z = 0;
  HxHyLz_n[i+j*dxNext+k*dxNext*dyNext] += -0.125*z;

  //HLH
  if (j>0)
    y = HxLyHz_df2[i+(j-1)*dxNext+k*dxNext*dyNext];
  else 
    y = 0;
  HxLyHz_n[i+j*dxNext+k*dxNext*dyNext] += -0.125*y;
      
  //LHH
  if (i>0)
    x = LxHyHz_df2[(i-1)+j*dxNext+k*dxNext*dyNext];
  else 
    x = 0;
  LxHyHz_n[i+j*dxNext+k*dxNext*dyNext] += -0.125*x;
}

extern "C" __global__ void cu_fwt3df_LC3(_data_t* HxHyHz_df1,_data_t* HxHyHz_df2,_data_t* HxHyHz_n,int dxNext, int dyNext, int dzNext)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t x,y,z;
  if ((i>=dxNext)||(j>=dyNext)||(k>=dzNext))
    {
      return;
    }

  //HHH
  x = HxHyHz_df1[i+j*dxNext+k*dxNext*dyNext];
  y = HxHyHz_df2[i+j*dxNext+k*dxNext*dyNext];
  z = HxHyHz_n[i+j*dxNext+k*dxNext*dyNext];
  HxHyHz_df1[i+j*dxNext+k*dxNext*dyNext] = 1.0/3.0*(-2.0*x+y+z);
  HxHyHz_df2[i+j*dxNext+k*dxNext*dyNext] = 1.0/3.0*(-x+2*y-z);
  HxHyHz_n[i+j*dxNext+k*dxNext*dyNext] = 1.0/3.0*(x+y+z);
}

// ############################################################################
// CUDA function of iwt depth convolution.
// Loads data to scratchpad (shared memory) and convolve w/ low pass and high pass
// Scratchpad size: K x 2*dy
// Output: Lz/Hz
// Input:  LxLy,LxHy / HxLy, HxHy, dx, dy, dxNext, dyNext,xOffset, yOffset,lod, hid, filterLen
// ############################################################################
extern "C" __global__ void cu_iwt3df_dep(_data_t *out, _data_t *Lz, _data_t *Hz, int dx, int dy,int dz,int dxNext, int dyNext, int dzNext,int xOffset, int yOffset,int zOffset,scalar_t *lod, scalar_t *hid, int filterLen)
{
  extern __shared__ _data_t deps [];
  int const K = blockDim.x;

  int ti = threadIdx.x;
  int tk = threadIdx.z;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  if (i>=dx){
    return;
  }
  for (int k = tk; k < dz; k += blockDim.z){
    deps[ti + k*K] = Lz[i + j*dx + k*dx*dy];
    deps[ti + (k+dz)*K] = Hz[i + j*dx + k*dx*dy];
  }
  __syncthreads();

  // Low-Pass and High Pass Downsample
  int ind;
  for (int k = tk+zOffset; k < dzNext+zOffset; k += blockDim.z){
	
    _data_t y = deps[0]-deps[0];
#pragma unroll
    for (int f = (k-(filterLen-1)) & 1; f < filterLen; f+=2){
      ind = (k-(filterLen-1)+f)>>1;
      if ((ind >= 0) && (ind < dz)) {
	y += deps[ti + ind*K] * lod[filterLen-1-f];
	y += deps[ti + (ind+dz)*K] * hid[filterLen-1-f];
      }
    }
	
    out[i + j*dx + (k-zOffset)*dx*dy] = y;
  }
}

// ############################################################################
// CUDA function of iwt row convolution. Assumes fwt_col() has already been called.
// Loads data to scratchpad (shared memory) and convolve w/ low pass and high pass
// Scratchpad size: K x 2*dy
// Output: Lx/Hx
// Input:  LxLy,LxHy / HxLy, HxHy, dx, dy, dxNext, dyNext,xOffset, yOffset,lod, hid, filterLen
// ############################################################################
extern "C" __global__ void cu_iwt3df_row(_data_t *out, _data_t *Ly, _data_t *Hy, int dx, int dy,int dz,int dxNext, int dyNext,int dzNext,int xOffset, int yOffset, int zOffset,scalar_t *lod, scalar_t *hid, int filterLen)
{
  extern __shared__ _data_t rows [];
  int const K = blockDim.x;

  int ti = threadIdx.x;
  int tj = threadIdx.y;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  if (i>=dx){
    return;
  }
  for (int j = tj; j < dy; j += blockDim.y){
    rows[ti + j*K] = Ly[i + j*dx + k*dx*dy];
    rows[ti + (j+dy)*K] = Hy[i + j*dx + k*dx*dy];
  }
  __syncthreads();

  // Low-Pass and High Pass Downsample
  int ind;
  for (int j = tj+yOffset; j < dyNext+yOffset; j += blockDim.y){
	
    _data_t y = rows[0]-rows[0];
#pragma unroll
    for (int f = (j-(filterLen-1)) & 1; f < filterLen; f+=2){
      ind = (j-(filterLen-1)+f)>>1;
      if ((ind >= 0) && (ind < dy)) {
	y += rows[ti + ind*K] * lod[filterLen-1-f];
	y += rows[ti + (ind+dy)*K] * hid[filterLen-1-f];
      }
    }
	
    out[i + (j-yOffset)*dx + k*dx*dyNext] = y;
  }
}

// ############################################################################
// CUDA function of iwt column convolution
// Loads data to scratchpad (shared memory) and convolve w/ low pass and high pass
// Scratchpad size: 2*dx x K
// Output: out
// Input:  Lx, Hx, dx, dy, dxNext, dyNext, lod, hid, filterLen
// ############################################################################
extern "C" __global__ void cu_iwt3df_col(_data_t *out, _data_t *Lx, _data_t *Hx, int dx, int dy,int dz,int dxNext, int dyNext, int dzNext,int xOffset, int yOffset, int zOffset,scalar_t *lod, scalar_t *hid, int filterLen)
{
  extern __shared__ _data_t cols [];

  int ti = threadIdx.x;
  int tj = threadIdx.y;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  if (j>=dyNext){
    return;
  }
  int dx2 = 2*dx;
  // Load Input to Temp Array
  for (int i = ti; i < dx; i += blockDim.x){
    cols[i + tj*dx2] = Lx[i + j*dx + k*dx*dyNext];
    cols[dx+i + tj*dx2] = Hx[i + j*dx + k*dx*dyNext];
  }
  __syncthreads();

  // Low-Pass and High Pass Downsample
  int ind;
  for (int i = ti+xOffset; i < dxNext+xOffset; i += blockDim.x){
    _data_t y = cols[0]-cols[0];
#pragma unroll
    for (int f = (i-(filterLen-1)) & 1; f < filterLen; f+=2){
      ind = (i-(filterLen-1)+f)>>1;
      if (ind >= 0 && ind < dx) {
	y += cols[ind + tj*dx2] * lod[filterLen-1-f];
	y += cols[dx+ind + tj*dx2] * hid[filterLen-1-f];
      }
    }
    out[(i-xOffset) + j*dxNext + k*dxNext*dyNext] = y;
  }
}

extern "C" __global__ void cu_iwt3df_LC1 (_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dx, int dy, int dz)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t df1,df2,n,xGreatZero,yGreatZero,zGreatZero;
  if ((i>=dx)||(j>=dy)||(k>=dz))
    {
      return;
    }

  //HLL
  df1 = HxLyLz_df1[i+j*dx+k*dx*dy];
  df2 = HxLyLz_df2[i+j*dx+k*dx*dy];
  n = HxLyLz_n[i+j*dx+k*dx*dy];
  HxLyLz_df2[i+j*dx+k*dx*dy] = df1;
  HxLyLz_n[i+j*dx+k*dx*dy] = df2;
  yGreatZero = j>0;
  zGreatZero = k>0;
  HxLyLz_df1[i+j*dx+k*dx*dy] = n - yGreatZero*0.25*df1 - zGreatZero*0.25*df2;

  //LHL
  df1 = LxHyLz_df1[i+j*dx+k*dx*dy];
  df2 = LxHyLz_df2[i+j*dx+k*dx*dy];
  n = LxHyLz_n[i+j*dx+k*dx*dy];
  LxHyLz_n[i+j*dx+k*dx*dy] = df2;
  xGreatZero = i>0;
  zGreatZero = k>0;
  LxHyLz_df2[i+j*dx+k*dx*dy] = n - xGreatZero*0.25*df1 - zGreatZero*0.25*df2;
      
  //LLH
  df1 = LxLyHz_df1[i+j*dx+k*dx*dy];
  df2 = LxLyHz_df2[i+j*dx+k*dx*dy];
  n = LxLyHz_n[i+j*dx+k*dx*dy];
  LxLyHz_df1[i+j*dx+k*dx*dy] = df2;
  LxLyHz_df2[i+j*dx+k*dx*dy] = df1;
  yGreatZero = j>0;
  xGreatZero = i>0;
  LxLyHz_n[i+j*dx+k*dx*dy] = n - yGreatZero*0.25*df1 - xGreatZero*0.25*df2;
}

extern "C" __global__ void cu_iwt3df_LC1_diff (_data_t *HxLyLz_df1,_data_t *HxLyLz_df2,_data_t *HxLyLz_n,_data_t *LxHyLz_df1,_data_t *LxHyLz_df2,_data_t *LxHyLz_n,_data_t *LxLyHz_df1,_data_t *LxLyHz_df2,_data_t *LxLyHz_n,int dx, int dy, int dz)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t x,y,z;
  if ((i>=dx)||(j>=dy)||(k>=dz))
    {
      return;
    }

  //HLL
  if (j>0)
    y = HxLyLz_df2[i+(j-1)*dx+k*dx*dy];
  else
    y = 0;
  if (k>0)
    z = HxLyLz_n[i+j*dx+(k-1)*dx*dy];
  else
    z = 0;
  HxLyLz_df1[i+j*dx+k*dx*dy] += 0.25*y + 0.25*z;

  //LHL
  if (i>0)
    x = LxHyLz_df1[(i-1)+j*dx+k*dx*dy];
  else
    x = 0;
  if (k>0)
    z = LxHyLz_n[i+j*dx+(k-1)*dx*dy];
  else
    z = 0;
  LxHyLz_df2[i+j*dx+k*dx*dy] += 0.25*x + 0.25*z;

  //LLH
  if (j>0)
    y = LxLyHz_df2[i+(j-1)*dx+k*dx*dy];
  else
    y = 0;
  if (i>0)
    x = LxLyHz_df1[(i-1)+j*dx+k*dx*dy];
  else
    x = 0;
  LxLyHz_n[i+j*dx+k*dx*dy] += 0.25*y + 0.25*x;
}

extern "C" __global__ void cu_iwt3df_LC2 (_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dx, int dy, int dz)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t df1,df2,n,xGreatZero,yGreatZero,zGreatZero;
  if ((i>=dx)||(j>=dy)||(k>=dz))
    {
      return;
    }

  //HHL
  df1 = HxHyLz_df1[i+j*dx+k*dx*dy];
  df2 = HxHyLz_df2[i+j*dx+k*dx*dy];
  n = HxHyLz_n[i+j*dx+k*dx*dy];
  HxHyLz_n[i+j*dx+k*dx*dy] = df2;
  zGreatZero = k>0;
  HxHyLz_df1[i+j*dx+k*dx*dy] = df1+n-zGreatZero*0.125*df2;
  HxHyLz_df2[i+j*dx+k*dx*dy] = -df1+n-zGreatZero*0.125*df2;

  //HLH
  df1 = HxLyHz_df1[i+j*dx+k*dx*dy];
  df2 = HxLyHz_df2[i+j*dx+k*dx*dy];
  n = HxLyHz_n[i+j*dx+k*dx*dy];
  HxLyHz_df2[i+j*dx+k*dx*dy] = df2;
  yGreatZero = j>0;
  HxLyHz_n[i+j*dx+k*dx*dy] = df1+n-yGreatZero*0.125*df2;
  HxLyHz_df1[i+j*dx+k*dx*dy] = -df1+n-yGreatZero*0.125*df2;
      
  //LHH
  df1 = LxHyHz_df1[i+j*dx+k*dx*dy];
  df2 = LxHyHz_df2[i+j*dx+k*dx*dy];
  n = LxHyHz_n[i+j*dx+k*dx*dy];
  LxHyHz_df1[i+j*dx+k*dx*dy] = df2;
  xGreatZero = i>0;
  LxHyHz_df2[i+j*dx+k*dx*dy] = df1+n-xGreatZero*0.125*df2;
  LxHyHz_n[i+j*dx+k*dx*dy] = -df1+n-xGreatZero*0.125*df2;
}

extern "C" __global__ void cu_iwt3df_LC2_diff (_data_t* HxHyLz_df1,_data_t* HxHyLz_df2,_data_t* HxHyLz_n,_data_t* HxLyHz_df1,_data_t* HxLyHz_df2,_data_t* HxLyHz_n,_data_t* LxHyHz_df1,_data_t* LxHyHz_df2,_data_t* LxHyHz_n,int dx, int dy, int dz)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t x,y,z;
  if ((i>=dx)||(j>=dy)||(k>=dz))
    {
      return;
    }

  //HHL
  if (k>0)
    z = HxHyLz_n[i+j*dx+(k-1)*dx*dy];
  else 
    z = 0;
  HxHyLz_df1[i+j*dx+k*dx*dy] += 0.125*z;
  HxHyLz_df2[i+j*dx+k*dx*dy] += 0.125*z;

  //HLH
  if (j>0)
    y = HxLyHz_df2[i+(j-1)*dx+k*dx*dy];
  else 
    y = 0;
  HxLyHz_df1[i+j*dx+k*dx*dy] += 0.125*y;
  HxLyHz_n[i+j*dx+k*dx*dy] += 0.125*y;
      
  //LHH
  if (i>0)
    x = LxHyHz_df1[(i-1)+j*dx+k*dx*dy];
  else 
    x = 0;
  LxHyHz_df2[i+j*dx+k*dx*dy] += 0.125*x;
  LxHyHz_n[i+j*dx+k*dx*dy] += 0.125*x;
}

extern "C" __global__ void cu_iwt3df_LC3 (_data_t* HxHyHz_df1,_data_t* HxHyHz_df2,_data_t* HxHyHz_n,int dx, int dy, int dz)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  int k = blockIdx.z*blockDim.z+threadIdx.z;
  _data_t df1,df2,n;
  if ((i>=dx)||(j>=dy)||(k>=dz))
    {
      return;
    }

  //HHH
  df1 = HxHyHz_df1[i+j*dx+k*dx*dy];
  df2 = HxHyHz_df2[i+j*dx+k*dx*dy];
  n = HxHyHz_n[i+j*dx+k*dx*dy];
  HxHyHz_df1[i+j*dx+k*dx*dy] = -df1+n;
  HxHyHz_df2[i+j*dx+k*dx*dy] = df2+n;
  HxHyHz_n[i+j*dx+k*dx*dy] = df1-df2+n;
}
extern "C" __global__ void cu_mult(_data_t* in, _data_t mult, int maxInd)
{
  int ind = blockIdx.x*blockDim.x+threadIdx.x;
  if (ind > maxInd)
    {
      return;
    }
  in[ind] = in[ind]*mult;
}

extern "C" __global__ void cu_add(_data_t* out, _data_t* in, int maxInd)
{
  int ind = blockIdx.x*blockDim.x+threadIdx.x;
  if (ind > maxInd)
    {
      return;
    }
  out[ind] += in[ind];
}

extern "C" __global__ void cu_add_mult(_data_t* out, _data_t* in, _data_t mult, int maxInd)
{
  int ind = blockIdx.x*blockDim.x+threadIdx.x;
  if (ind > maxInd)
    {
      return;
    }
  _data_t i = out[ind];
  out[ind] = i+(out[ind]-i)*mult;
}

__global__ void cu_soft_thresh (_data_t* in, scalar_t thresh, int numMax)
{
  int const i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i>numMax)
    return;
  scalar_t norm = abs(in[i]);
  scalar_t red = norm - thresh;
  in[i] = (red > 0.f) ? ((red / norm) * (in[i])) : in[i]-in[i];
}

__global__ void cu_circshift(_data_t* data, _data_t* dataCopy, int dx, int dy, int dz,int shift1, int shift2,int shift3) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if (index >= dx*dy*dz) {
    return;
  }
  int indexShifted = (index+shift1+shift2*dx+shift3*dx*dy)%(dx*dy*dz);
  data[indexShifted] = dataCopy[index];
}

__global__ void cu_circunshift(_data_t* data, _data_t* dataCopy, int dx, int dy, int dz,int shift1, int shift2,int shift3) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if (index >= dx*dy*dz) {
    return;
  }
  int indexShifted = (index+shift1+shift2*dx+shift3*dx*dy)%(dx*dy*dz);
  data[index] = dataCopy[indexShifted];
}

